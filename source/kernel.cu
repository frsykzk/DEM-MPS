#include "hip/hip_runtime.h"
#include "class.cuh"

void DEMPS::RdDat() {
	//////////////////////physical.txt///////////////////////////////
	FILE* in0;

	if (fopen_s(&in0, IN_FILE_0, "r") != 0) {
		printf("physical.txt���J���܂���\n");
	}
	else {
		real scan[50];
		fscanf_s(in0, "%f %f %f", &scan[0], &scan[1], &scan[2]);//�ŏ��͈�
		fscanf_s(in0, "%f %f %f", &scan[3], &scan[4], &scan[5]);//�ő�͈�
		fscanf_s(in0, "%f %f %f", &scan[6], &scan[7], &scan[8]);//�d�͉����x
		fscanf_s(in0, "%f", &scan[9]);//�v�Z�I������
		fscanf_s(in0, "%f", &scan[10]);//�t�@�C���o�͊Ԋu
		fscanf_s(in0, "%f", &scan[11]);//�Ǘ��q���a
		fclose(in0);

		WLL_PCL_DST = real(scan[11]);
		MINc.x = real(scan[0] - 3.1f * WLL_PCL_DST);
		MINc.y = real(scan[1] - 3.1f * WLL_PCL_DST);
		MINc.z = real(scan[2] - 3.1f * WLL_PCL_DST);
		MAXc.x = real(scan[3] + 3.1f * WLL_PCL_DST);
		MAXc.y = real(scan[4] + 3.1f * WLL_PCL_DST);
		MAXc.z = real(scan[5] + 3.1f * WLL_PCL_DST);
		G.x = real(scan[6]);
		G.y = real(scan[7]);
		G.z = real(scan[8]);
		FIN_TIM = real(scan[9]);
		output_time = real(scan[10]);

		printf_s("MINc.x = %f  MINc.y = %f  MINc.z = %f\n", MINc.x, MINc.y, MINc.z);
		printf_s("MAXc.x = %f  MAXc.y = %f  MAXc.z = %f\n", MAXc.x, MAXc.y, MAXc.z);
		printf_s("G.x = %f  G.y = %f  G.z = %f\n", G.x, G.y, G.z);
		printf_s("FIN_TIM = %f\n", FIN_TIM);
		printf_s("output_time = %f\n", output_time);
		printf_s("WLL_PCL_DST = %f\n\n", WLL_PCL_DST);

	}
	//////////////////////MPS_physical.txt///////////////////////////////


	//////////////////////DEM_physical.txt///////////////////////////////
	FILE* in1;

	if (fopen_s(&in1, IN_FILE_1, "r") != 0) {
		printf("DEM_physical.txt���J���܂���\n");
	}
	else {
		real scan[50];
		fscanf_s(in1, "%f", &scan[0]);//���ԍ��ݕ�
		fscanf_s(in1, "%f", &scan[1]);//�΂˒萔
		fscanf_s(in1, "%f", &scan[2]);//���C�W��
		fscanf_s(in1, "%f", &scan[3]);//�ő̗��q���a
		fclose(in1);

		DEMdt = real(scan[0]);
		k = real(scan[1]);
		mu = real(scan[2]);
		SLD_PCL_DST = real(scan[3]);

		printf("DEMdt = %f\n", DEMdt);
		printf("k = %f\n", k);
		printf("mu = %f\n", mu);
		printf("SLD_PCL_DST = %f\n\n", SLD_PCL_DST);
	}
	//////////////////////DEM_physical.txt///////////////////////////////


	//////////////////////MPS_physical.txt///////////////////////////////
	FILE* in2;

	if (fopen_s(&in2, IN_FILE_2, "r") != 0) {
		printf("MPS_physical.txt���J���܂���\n");
	}
	else {
		real scan[50];
		fscanf_s(in2, "%f", &scan[0]);//�}�b�n��
		fscanf_s(in2, "%f", &scan[1]);//�N�[������
		fscanf_s(in2, "%f", &scan[2]);//�����W��
		fscanf_s(in2, "%f", &scan[3]);//���E�ڋߋ���
		fscanf_s(in2, "%f", &scan[4]);//���S���W��
		fscanf_s(in2, "%f", &scan[5]);//���̗��q���a
		fclose(in2);

		Ma = real(scan[0]);
		CRT_NUM = real(scan[1]);
		COL_RAT = real(scan[2]);
		DST_LMT_RAT = real(scan[3]);
		KNM_VSC = real(scan[4]);
		FLD_PCL_DST = real(scan[5]);

		printf("Ma = %f\n", Ma);
		printf("CRT_NUM = %f\n", CRT_NUM);
		printf("COL_RAT= %f\n", COL_RAT);
		printf("DST_LMT_RAT = %f\n", DST_LMT_RAT);
		printf("KNM_VSC  = %f\n", KNM_VSC);
		printf("FLD_PCL_DST = %f\n\n", FLD_PCL_DST);
	}
	//////////////////////MPS_physical.txt///////////////////////////////



	//////////////////////initial.txt///////////////////////////////
	FILE* in3;
	if (fopen_s(&in3, IN_FILE_3, "r") != 0) {
		printf_s("initial.txt���J���܂���\n");
	}
	else {
		fscanf_s(in3, "%d", &nP);//�����q���擾
		std::cout << "�����q��(�t�@�C���擪�l) nP = " << nP << std::endl;

		//���q
		Pos.x = (real*)malloc(sizeof(real) * (nP));
		Pos.y = (real*)malloc(sizeof(real) * (nP));
		Pos.z = (real*)malloc(sizeof(real) * (nP));

		Vel.x = (real*)malloc(sizeof(real) * (nP));
		Vel.y = (real*)malloc(sizeof(real) * (nP));
		Vel.z = (real*)malloc(sizeof(real) * (nP));

		Omega.x = (real*)malloc(sizeof(real) * (nP));
		Omega.y = (real*)malloc(sizeof(real) * (nP));
		Omega.z = (real*)malloc(sizeof(real) * (nP));

		Ftotal.x = (real*)malloc(sizeof(real) * (nP));
		Ftotal.y = (real*)malloc(sizeof(real) * (nP));
		Ftotal.z = (real*)malloc(sizeof(real) * (nP));

		Torque.x = (real*)malloc(sizeof(real) * (nP));
		Torque.y = (real*)malloc(sizeof(real) * (nP));
		Torque.z = (real*)malloc(sizeof(real) * (nP));

		Acc.x = (real*)malloc(sizeof(real) * (nP));
		Acc.y = (real*)malloc(sizeof(real) * (nP));
		Acc.z = (real*)malloc(sizeof(real) * (nP));

		Prs = (real*)malloc(sizeof(real) * (nP));
		pav = (real*)malloc(sizeof(real) * (nP));

		D = (real*)malloc(sizeof(real) * (nP));
		Typ = (char*)malloc(sizeof(char) * (nP));
		Dns = (real*)malloc(sizeof(real) * (Dns_Num));
		//���q


		//��
		WLLVec.x = (real*)malloc(sizeof(real) * (nP));
		WLLVec.y = (real*)malloc(sizeof(real) * (nP));
		WLLVec.z = (real*)malloc(sizeof(real) * (nP));
		WLLSE = (char*)malloc(sizeof(char) * (nP));
		//��

		//�~���[
		PosM.x = (real*)malloc(sizeof(real) * (nP * NumMRR));
		PosM.y = (real*)malloc(sizeof(real) * (nP * NumMRR));
		PosM.z = (real*)malloc(sizeof(real) * (nP * NumMRR));
		VelM.x = (real*)malloc(sizeof(real) * (nP * NumMRR));
		VelM.y = (real*)malloc(sizeof(real) * (nP * NumMRR));
		VelM.z = (real*)malloc(sizeof(real) * (nP * NumMRR));
		PrsM = (real*)malloc(sizeof(real) * (nP * NumMRR));
		TypM = (char*)malloc(sizeof(char) * (nP * NumMRR));
		//�~���[


		int nPsolid = 0;
		int nPfluid = 0;
		int nPwall = 0;
		int nPobj = 0;
		int nPobj2 = 0;
		int nPtmp = 0;
		for (int i = 0; i < nP; i++) {
			int a[1];
			float b[11];
			int c[1];
			float g[1];
			fscanf_s(in3, " %d %d %f %f %f %f %f %f %f", &a[0], &c[0], &b[0], &b[1], &b[2], &b[8], &b[9], &b[10], &g[0]);
			const treal3 pos = { b[0], b[1], b[2] };
			if (pos.x<MAXc.x && pos.x>MINc.x && pos.y<MAXc.y && pos.y>MINc.y && pos.z<MAXc.z && pos.z>MINc.z) {
				Typ[nPtmp] = char(c[0]);
				Pos.x[nPtmp] = real(b[0]); Pos.y[nPtmp] = real(b[1]); Pos.z[nPtmp] = real(b[2]);
				Vel.x[nPtmp] = Vel.y[nPtmp] = Vel.z[nPtmp] = 0.0f;
				Acc.x[nPtmp] = Acc.y[nPtmp] = Acc.z[nPtmp] = 0.0f;
				Prs[nPtmp] = 0.0f;
				pav[nPtmp] = 0.0f;

				if (Typ[nPtmp] == SLD) { nPsolid += 1; WLLVec.x[nPtmp] = 0.0f; WLLVec.y[nPtmp] = 0.0f; WLLVec.z[nPtmp] = 0.0f; D[nPtmp] = SLD_PCL_DST; }
				else if (Typ[nPtmp] == FLD) { nPfluid += 1; WLLVec.x[nPtmp] = 0.0f; WLLVec.y[nPtmp] = 0.0f; WLLVec.z[nPtmp] = 0.0f;  D[nPtmp] = FLD_PCL_DST; }
				else if (Typ[nPtmp] == WLL) { nPwall += 1; WLLVec.x[nPtmp] = real(b[8]); WLLVec.y[nPtmp] = real(b[9]); WLLVec.z[nPtmp] = real(b[10]); D[nPtmp] = WLL_PCL_DST; }
				else if (Typ[nPtmp] == OBJ) { nPobj += 1; WLLVec.x[nPtmp] = real(b[8]); WLLVec.y[nPtmp] = real(b[9]); WLLVec.z[nPtmp] = real(b[10]);  D[nPtmp] = WLL_PCL_DST; }
				else if (Typ[nPtmp] == OBJ2) { nPobj2 += 1; WLLVec.x[nPtmp] = real(b[8]); WLLVec.y[nPtmp] = real(b[9]); WLLVec.z[nPtmp] = real(b[10]);  D[nPtmp] = WLL_PCL_DST; }
				nPtmp += 1;
			}
		}
		nP = nPtmp;
		nPWLL = nPwall;
		nPSLD = nPsolid;
		nPFLD = nPfluid;
		nPOBJ = nPobj;
		nPOBJ2 = nPobj2;

		ep.x = (real*)malloc(sizeof(real) * (NCP * nPSLD));
		ep.y = (real*)malloc(sizeof(real) * (NCP * nPSLD));
		ep.z = (real*)malloc(sizeof(real) * (NCP * nPSLD));
		pair = (int*)malloc(sizeof(int) * (NCP * nPSLD));

		std::cout << "���ő̗��q�� nPSLD = " << nPSLD << std::endl;
		std::cout << "�����̗��q�� nPFLD = " << nPFLD << std::endl;
		std::cout << "���Ǘ��q�� nPWLL = " << nPWLL << std::endl;
		std::cout << "�����Ǘ��q�� nPOBJ = " << nPOBJ << std::endl;
		std::cout << "�����Ǔ󗱎q�� nPOBJ2 = " << nPOBJ2 << std::endl;
		std::cout << "�����q�� nP = " << nP << std::endl;
	}
	fclose(in3);
	//////////////////////initial.txt///////////////////////////////
}


__global__ void d_initialize_int_array(const int n, int* i_array, const int a) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) { return; }
	i_array[i] = a;
}

__global__ void d_initialize_real_array(const int n, real* i_array, real a) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) { return; }
	i_array[i] = a;
}


void DEMPS::Output(const char typ, const int outputflg) {
	if (outputflg == 0)//�ő̗��q�o��
	{
		sprintf(outout_filename, "./output/outputSLD%05d.csv", iF);
		printf("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "Pos.x,Pos.y,Pos.z,V.x,V.y,V.z,Vel,F.x,F.y,F.z,Force\n");
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf(fp, "%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f\n", Pos.x[i], Pos.y[i], Pos.z[i],
						Vel.x[i], Vel.y[i], Vel.z[i], sqrt(Vel.x[i] * Vel.x[i] + Vel.y[i] * Vel.y[i] + Vel.z[i] * Vel.z[i]),
						Ftotal.x[i], Ftotal.y[i], Ftotal.z[i], sqrt(Ftotal.x[i] * Ftotal.x[i] + Ftotal.y[i] * Ftotal.y[i] + Ftotal.z[i] * Ftotal.z[i]));
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 1)//���̗��q�o��
	{
		sprintf_s(outout_filename, "./output/outputFLD%05d.csv", iF);
		printf_s("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "Pos.x,Pos.y,Pos.z,Vel.x,Vel.y,Vel.z,Vel,Prs,pav\n");
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf_s(fp, "%f,%f,%f,%f,%f,%f,%f,%f,%f\n", Pos.x[i], Pos.y[i], Pos.z[i], Vel.x[i], Vel.y[i], Vel.z[i], sqrt(Vel.x[i] * Vel.x[i] + Vel.y[i] * Vel.y[i] + Vel.z[i] * Vel.z[i]), Prs[i], pav[i] / OPT_FQC);
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 2)//�Ǘ��q�o�́@�����I�ɂ͕ǂɓ�����(�ő̏Փ˗́E������)���o�͂�����
	{
		sprintf(outout_filename, "./output/outputWLL%05d.csv", iF);
		printf("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "Pos.x,Pos.y,Pos.z,Prs,pav,SE\n");
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf_s(fp, "%f,%f,%f,%f,%f,%d\n", Pos.x[i], Pos.y[i], Pos.z[i], Prs[i], pav[i] / OPT_FQC, WLLSE[i]);
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 3)//���Ǘ��q�o�́@�����I�ɂ͕ǂɓ�����(�ő̏Փ˗́E������)���o�͂�����
	{
		sprintf(outout_filename, "./output/outputOBJ%05d.csv", iF);
		printf("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "Pos.x,Pos.y,Pos.z,Prs,pav,SE\n");
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf_s(fp, "%f,%f,%f,%f,%f,%d\n", Pos.x[i], Pos.y[i], Pos.z[i], Prs[i], pav[i] / OPT_FQC, WLLSE[i]);
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 4)//���Ǘ��q��o�́@�����I�ɂ͕ǂɓ�����(�ő̏Փ˗́E������)���o�͂�����
	{
		sprintf(outout_filename, "./output/outputOBJ2%05d.csv", iF);
		printf("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "Pos.x,Pos.y,Pos.z,Prs,pav,SE\n");
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf_s(fp, "%f,%f,%f,%f,%f,%d\n", Pos.x[i], Pos.y[i], Pos.z[i], Prs[i], pav[i] / OPT_FQC, WLLSE[i]);
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 5)//�~���[���q�o��
	{
		sprintf_s(outout_filename, "./output/outputMRR%05d.csv", iF);
		printf_s("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf_s("%s���J���܂���\n", outout_filename);
		}
		else {
			fprintf_s(fp, "PosM.x,PosM.y,PosM.z,VelM,PrsM\n");
			for (int i = 0; i < nP; i++) {
				for (int k = 0; k < NumMRR; k++) {
					int kiNM = k + i * NumMRR;
					if (TypM[kiNM] == typ) {
						fprintf_s(fp, "%f,%f,%f,%f,%f,\n", PosM.x[kiNM], PosM.y[kiNM], PosM.z[kiNM], sqrt(VelM.x[kiNM] * VelM.x[kiNM] + VelM.y[kiNM] * VelM.y[kiNM] + VelM.z[kiNM] * VelM.z[kiNM]), PrsM[kiNM]);
					}
				}
			}
		}
		fclose(fp);
	}

}


void DEMPS::Output2(const char typ, const int outputflg) {//seitei.txt�o��
	if (outputflg == 0)//SLD_seitei
	{
		sprintf(outout_filename, "./output/DEM_seitei.txt");
		printf("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf("%s���J���܂���\n", outout_filename);
		}
		else {
			int sldnp = 0;
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) { sldnp += 1; }//�ő̗��q�J�E���g
			}
			fprintf(fp, "%d\n", sldnp);
			int SLDnP = 0;
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf(fp, "%d %d %f %f %f %f %f %f %f\n", SLDnP, Typ[i], Pos.x[i], Pos.y[i], Pos.z[i], 0.0f, 0.0f, 0.0f, 0.0f);
					SLDnP += 1;
				}
			}
		}
		fclose(fp);
	}

	else if (outputflg == 1)//FLD_seitei
	{
		sprintf_s(outout_filename, "./output/MPS_seitei.txt");
		printf_s("Filename = %s\n", outout_filename);

		if (fopen_s(&fp, outout_filename, "w") != 0) {
			printf_s("%s���J���܂���\n", outout_filename);
		}
		else {
			int fldnp = 0;
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) { fldnp += 1; }//���̗��q�J�E���g
			}
			fprintf_s(fp, "%d\n", fldnp);
			int FLDnP = 0;
			for (int i = 0; i < nP; i++) {
				if (Typ[i] == typ) {
					fprintf_s(fp, "%d %d %f %f %f %f %f %f %f\n", FLDnP, Typ[i], Pos.x[i], Pos.y[i], Pos.z[i], 0.0f, 0.0f, 0.0f, 0.0f);
					FLDnP += 1;
				}
			}
		}
		fclose(fp);
	}

}


void DEMPS::WrtDat(void) {
#if DEM_flg
	Output(SLD, 0);//�ő̗��q�o��
#endif

#if MPS_flg
	Output(FLD, 1);//�����q�o��
	Output(MRR, 5);//�����~���[�m�F
#endif

#if move_WLL
	Output(WLL, 2);//�����q�o��
	Output(OBJ, 3);//�����q�o��
	Output(OBJ2, 4);//�����q�o��
#endif

	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	((d_initialize_real_array << <blocks_nP, threads >> > (nP, d_pav, 0.0f)));//���ψ��͏�����
	//DEM�̗͂����ԕ��ςƂ肽��
	//CHECK(hipDeviceSynchronize());
	printf_s("WrtDat finished!\n\n");
}


void DEMPS::WrtDatWLL(void) {
	Output(WLL, 2);
	printf("WrtDatWLL finished!\n\n");
}


void DEMPS::WrtDat2(void) {//seitei.txt
	Output2(SLD, 0);
	Output2(FLD, 1);
	printf_s("WrtDat2 finished!\n\n");
}


void DEMPS::AlcBkt() {

	PCL_DST = WLL_PCL_DST;
	if (SLD_PCL_DST > PCL_DST) { PCL_DST = SLD_PCL_DST; }
	if (FLD_PCL_DST > PCL_DST) { PCL_DST = FLD_PCL_DST; }//�ł������ɍ��킹�ăo�P�b�g���

	r = FLD_PCL_DST * 3.1f;
	r2 = r * r;
	rp = FLD_PCL_DST * 2.1f;//���͗p
	rp2 = rp * rp;

	DB = PCL_DST * 3.1f;
	DB2 = DB * DB;
	DBinv = 1.0f / DB;

	nBx = (int)((MAXc.x - MINc.x) * DBinv) + 3;
	nBy = (int)((MAXc.y - MINc.y) * DBinv) + 3;
	nBz = (int)((MAXc.z - MINc.z) * DBinv) + 3;

	nBxy = nBx * nBy;
	nBxyz = nBx * nBy * nBz;
	printf_s("nBx:%d  nBy:%d  nBz:%d  nBxy:%d  nBxyz:%d\n", nBx, nBy, nBz, nBxy, nBxyz);

	(hipMalloc((void**)&d_bfst, sizeof(int) * nBxyz));
	(hipMalloc((void**)&d_blst, sizeof(int) * nBxyz));
	(hipMalloc((void**)&d_nxt, sizeof(int) * (nP)));
	(hipMalloc((void**)&d_bfstM, sizeof(int) * nBxyz));
	(hipMalloc((void**)&d_blstM, sizeof(int) * nBxyz));
	(hipMalloc((void**)&d_nxtM, sizeof(int) * (nP * NumMRR)));
}


void DEMPS::SetPara() {
	///////////////////////////////////////////////////////����///////////////////////////////////////////////////////////////////

	//�����ƃ^�C���X�e�b�v
	real max_height = MINc.y;
	real min_height = MAXc.y;
	for (int i = 0; i < nP; i++) {//������
		if (Typ[i] == FLD) {
			if (max_height < Pos.y[i]) { max_height = Pos.y[i]; }
			else if (min_height > Pos.y[i]) { min_height = Pos.y[i]; }
		}
	}
	real lmm = max_height - min_height;
	ulmax = sqrt(2.0f * abs(G.y) * lmm);

	real max_height2 = MINc.y;
	real min_height2 = MAXc.y;
	for (int i = 0; i < nP; i++) {//������
		if (Typ[i] == SLD) {
			if (max_height2 < Pos.y[i]) { max_height2 = Pos.y[i]; }
			else if (min_height2 > Pos.y[i]) { min_height2 = Pos.y[i]; }
		}
	}
	usmax = sqrt(2.0f * abs(G.y) * (max_height2 - min_height2));


#pragma omp parallel for
	for (int i = 0; i < Dns_Num; i++) {//������
		Dns[i] = 0.0f;
	}
	Dns[SLD] = Dns_SLD;
	Dns[FLD] = Dns_FLD;
	Dns[WLL] = Dns_WLL;
	Dns[OBJ] = Dns_OBJ;
	Dns[OBJ2] = Dns_OBJ2;
	Dns[MRR] = Dns_FLD;

#pragma omp parallel for
	for (int i = 0; i < nP; i++) {
		Ftotal.x[i] = Ftotal.y[i] = Ftotal.z[i] = 0.0f;
		Omega.x[i] = Omega.y[i] = Omega.z[i] = 0.0f;
		Torque.x[i] = Torque.y[i] = Torque.z[i] = 0.0f;

		if (Typ[i] == SLD) { D[i] = SLD_PCL_DST; }
		else if (Typ[i] == FLD) { D[i] = FLD_PCL_DST; }
		else if (Typ[i] == WLL) { D[i] = WLL_PCL_DST; }
		else if (Typ[i] == OBJ) { D[i] = WLL_PCL_DST; }
		else if (Typ[i] == OBJ2) { D[i] = WLL_PCL_DST; }
	}

	Vol_SLD = pi * SLD_PCL_DST * SLD_PCL_DST * SLD_PCL_DST / 6.0f;
	Vol_FLD = pi * FLD_PCL_DST * FLD_PCL_DST * FLD_PCL_DST / 6.0f;

	///////////////////////////////////////////////////////����///////////////////////////////////////////////////////////////////


	///////////////////////////////////////////////////////DEM///////////////////////////////////////////////////////////////////

	m = Dns[SLD] * pi * SLD_PCL_DST * SLD_PCL_DST * SLD_PCL_DST / 6.0f;

	I = 0.1f * m * SLD_PCL_DST * SLD_PCL_DST;

#if 1
	real alpha = 0.27;

	kn = k;
	kt = kn / (2.0f * (1.0f + alpha));

	eta_n = 2.0f * sqrt(m * kn);
	eta_t = eta_n / sqrt(2.0f * (1.0f + alpha));
#else
	kn = 1000;
	kt = kn * 0.25f;

	eta_n = sqrt(2.0f * m * kn);
	eta_t = sqrt(2.0f * m * kt);

#endif


#pragma omp parallel for
	for (int i = 0; i < nPSLD; i++) { for (int k = 0; k < NCP; k++) { ep.x[k + i * NCP] = 0.0f; } }
#pragma omp parallel for
	for (int i = 0; i < nPSLD; i++) { for (int k = 0; k < NCP; k++) { ep.y[k + i * NCP] = 0.0f; } }
#pragma omp parallel for
	for (int i = 0; i < nPSLD; i++) { for (int k = 0; k < NCP; k++) { ep.z[k + i * NCP] = 0.0f; } }
#pragma omp parallel for
	for (int i = 0; i < nPSLD; i++) { for (int k = 0; k < NCP; k++) { pair[k + i * NCP] = -2; } }

	printf("m:%.10f\nkn:%f  kt:%f\neta_n:%f  eta_t:%f\n\n", m, kn, kt, eta_n, eta_t);

	///////////////////////////////////////////////////////DEM///////////////////////////////////////////////////////////////////	



	///////////////////////////////////////////////////////MPS///////////////////////////////////////////////////////////////////

	//�������q���x
	real tn0 = 0.0f;
	real tn0_grad = 0.0f;
	real tlmd = 0.0f;
	real tlmd_grad = 0.0f;

	for (int ix = -10; ix < 10; ix++) {
		for (int iy = -10; iy < 10; iy++) {
			for (int iz = -10; iz < 10; iz++) {
				real x = real(FLD_PCL_DST) * real(ix);
				real y = real(FLD_PCL_DST) * real(iy);
				real z = real(FLD_PCL_DST) * real(iz);
				real dist2 = x * x + y * y + z * z;
				if (dist2 == 0.0f) { continue; }
				real dist = sqrt(dist2);
				if (dist2 < rp2) {
					tn0_grad += WEI_grad(dist, rp);
					tlmd_grad += dist2 * WEI_grad(dist, rp);
				}
				if (dist2 < r2) {
					tn0 += WEI(dist, r);
					tlmd += dist2 * WEI(dist, r);
				}
			}
		}
	}
	n0 = tn0;
	lmd = tlmd / tn0;
	n0_grad = tn0_grad;
	lmd_grad = tlmd_grad / tn0_grad;
	printf_s("n0:%f\nn0_grad:%f\n", n0, n0_grad);

	//�ڋߋ֎~����
	rlim = FLD_PCL_DST * DST_LMT_RAT;
	rlim2 = rlim * rlim;

	COL = real(COL_RAT + 1.0f);//�����W���ݒ�

#pragma omp parallel for
	for (int i = 0; i < nP; i++) {//������
		for (int k = 0; k < NumMRR; k++) {
			int kiNM = k + i * NumMRR;
			PosM.x[kiNM] = PosM.y[kiNM] = PosM.z[kiNM] = 0.0f;
			VelM.x[kiNM] = VelM.y[kiNM] = VelM.z[kiNM] = 0.0f;
			PrsM[kiNM] = 0.0f;
			TypM[kiNM] = GST;
		}
		WLLSE[i] = Surface;
	}



	SND = ulmax / Ma;
	Prs_coef = SND * SND / n0_grad;
	Vsc_coef = 2.0f * 3.0f * KNM_VSC / n0 / lmd;
	Pmax = 1000.0f * abs(G.y) * lmm;//9.80665f 

	///////////////////////////////////////////////////////MPS///////////////////////////////////////////////////////////////////


	//dt = CRT_NUM * PCL_DST / SND;
	dt = 0.00001f;//�Œ�
	//dt = real(PCL_DST / (ulmax + SND));//�v�Z�Ԋu(�_���u���C�N�̍ő呬�x����ɂ���)
	if (DEMdt < dt) { dt = DEMdt; }

	printf_s("�^�C���X�e�b�v:dt=%f\n�ő嗬�̑��x:ulmax=%f\n�ő�ő̑��x:usmax=%f\n����=%f\nVsc_coef=%f\nPrs_coef=%f\nPmax=%f\n", dt, ulmax, usmax, SND, Vsc_coef, Prs_coef, Pmax);


	//�z��m�ۂ�VRAM�ւ̓]��
	(hipMalloc((void**)&d_Typ, sizeof(char) * nP));
	(hipMalloc((void**)&d_Pos.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Pos.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Pos.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_Vel.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Vel.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Vel.z, sizeof(real) * nP));

	(hipMemcpy(d_Typ, Typ, sizeof(char) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Pos.x, Pos.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Pos.y, Pos.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Pos.z, Pos.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Vel.x, Vel.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Vel.y, Vel.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Vel.z, Vel.z, sizeof(real) * nP, hipMemcpyHostToDevice));

	///////////////////////////////////////////////////////DEM///////////////////////////////////////////////////////////////////
	(hipMalloc((void**)&d_D, sizeof(real) * nP));
	(hipMalloc((void**)&d_Ftotal.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Ftotal.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Ftotal.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_Omega.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Omega.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Omega.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_Torque.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Torque.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Torque.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_ep.x, sizeof(real) * (NCP * nPSLD)));
	(hipMalloc((void**)&d_ep.y, sizeof(real) * (NCP * nPSLD)));
	(hipMalloc((void**)&d_ep.z, sizeof(real) * (NCP * nPSLD)));
	(hipMalloc((void**)&d_pair, sizeof(int) * (NCP * nPSLD)));


	(hipMemcpy(d_D, D, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Ftotal.x, Ftotal.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Ftotal.y, Ftotal.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Ftotal.z, Ftotal.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Omega.x, Omega.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Omega.y, Omega.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Omega.z, Omega.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Torque.x, Torque.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Torque.y, Torque.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Torque.z, Torque.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_ep.x, ep.x, sizeof(real) * (NCP * nPSLD), hipMemcpyHostToDevice));
	(hipMemcpy(d_ep.y, ep.y, sizeof(real) * (NCP * nPSLD), hipMemcpyHostToDevice));
	(hipMemcpy(d_ep.z, ep.z, sizeof(real) * (NCP * nPSLD), hipMemcpyHostToDevice));
	(hipMemcpy(d_pair, pair, sizeof(int) * (NCP * nPSLD), hipMemcpyHostToDevice));
	///////////////////////////////////////////////////////DEM///////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////MPS///////////////////////////////////////////////////////////////////
	(hipMalloc((void**)&d_Acc.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_Acc.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_Acc.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_Prs, sizeof(real) * nP));
	(hipMalloc((void**)&d_pav, sizeof(real) * nP));
	(hipMalloc((void**)&d_Dns, sizeof(real) * Dns_Num));

	(hipMalloc((void**)&d_WLLVec.x, sizeof(real) * nP));
	(hipMalloc((void**)&d_WLLVec.y, sizeof(real) * nP));
	(hipMalloc((void**)&d_WLLVec.z, sizeof(real) * nP));
	(hipMalloc((void**)&d_WLLSE, sizeof(char) * nP));

	(hipMalloc((void**)&d_FromWLL, sizeof(int) * (nP * NumMRR)));//�f�o�C�X�̂�(�]���Ȃ�)

	(hipMalloc((void**)&d_TypM, sizeof(char) * (nP * NumMRR)));
	(hipMalloc((void**)&d_PosM.x, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_PosM.y, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_PosM.z, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_VelM.x, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_VelM.y, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_VelM.z, sizeof(real) * (nP * NumMRR)));
	(hipMalloc((void**)&d_PrsM, sizeof(real) * (nP * NumMRR)));


	(hipMemcpy(d_Acc.x, Acc.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Acc.y, Acc.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Acc.z, Acc.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Prs, Prs, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_pav, pav, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_Dns, Dns, sizeof(real) * Dns_Num, hipMemcpyHostToDevice));

	(hipMemcpy(d_WLLVec.x, WLLVec.x, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_WLLVec.y, WLLVec.y, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_WLLVec.z, WLLVec.z, sizeof(real) * nP, hipMemcpyHostToDevice));
	(hipMemcpy(d_WLLSE, WLLSE, sizeof(char) * nP, hipMemcpyHostToDevice));

	(hipMemcpy(d_TypM, TypM, sizeof(char) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_PosM.x, PosM.x, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_PosM.y, PosM.y, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_PosM.z, PosM.z, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_VelM.x, VelM.x, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_VelM.y, VelM.y, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_VelM.z, VelM.z, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	(hipMemcpy(d_PrsM, PrsM, sizeof(real) * (nP * NumMRR), hipMemcpyHostToDevice));
	///////////////////////////////////////////////////////MPS///////////////////////////////////////////////////////////////////

}


__global__ void d_MkBkt(const int nP, const  int nBx, const  int nBxy, const  real DBinv, int* d_bfst, int* d_blst, int* d_nxt, const char* d_Typ, areal3 d_Pos, const treal3 MINc)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	if (d_Typ[i] == GST) { return; }

	int ix = (int)((d_Pos.x[i] - MINc.x) * DBinv) + 1;
	int iy = (int)((d_Pos.y[i] - MINc.y) * DBinv) + 1;
	int iz = (int)((d_Pos.z[i] - MINc.z) * DBinv) + 1;
	int ib = iz * nBxy + iy * nBx + ix;
	const int j = atomicExch(&d_blst[ib], i);
	if (j == -1) { d_bfst[ib] = i; }
	else { d_nxt[j] = i; }

}


void DEMPS::MkBkt() {//���q���o�P�b�g�Ɏ��[
	//printf_s("MkBkt start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_bfst, -1)));
	((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_blst, -1)));
	((d_initialize_int_array << <blocks_nP, threads >> > (nP, d_nxt, -1)));
	//CHECK(hipDeviceSynchronize());

	d_MkBkt << <blocks_nP, threads >> > (nP, nBx, nBxy, DBinv, d_bfst, d_blst, d_nxt, d_Typ, d_Pos, MINc);
	//CHECK(hipDeviceSynchronize());

	//printf_s("MkBkt finished!\n\n");
}


#if MPS_flg
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////MPS_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void d_Surface_Edge(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_WLLVec, char* d_WLLSE, const real WLL_PCL_DST,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)//�Ǘ��qsurface edge����
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	real r2 = (1.0f * WLL_PCL_DST) * (1.0f * WLL_PCL_DST);//�אڕǗ��q���苗����2��
	char Typ = d_Typ[i];

	if (Typ == WLL) {
		treal3 pos;
		pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
		treal3 vec;
		vec.x = d_WLLVec.x[i];	vec.y = d_WLLVec.y[i];	vec.z = d_WLLVec.z[i];

		int ix = (int)((d_Pos.x[i] - MINc.x) * DBinv) + 1;
		int iy = (int)((d_Pos.y[i] - MINc.y) * DBinv) + 1;
		int iz = (int)((d_Pos.z[i] - MINc.z) * DBinv) + 1;
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							if (d_Typ[j] == WLL) {
								treal3 p;//i,j�̋����̐���
								p.x = pos.x - d_Pos.x[j];	p.y = pos.y - d_Pos.y[j];	p.z = pos.z - d_Pos.z[j];
								real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
								if (dist2 < r2) {
									if ((d_WLLVec.x[j] != vec.x) || (d_WLLVec.y[j] != vec.y) || (d_WLLVec.z[j] != vec.z)) {//�߂��̗��q�Ɩ@���x�N�g���Ⴄ�Ȃ�edge
										d_WLLSE[i] = Edge;
									}
								}
							}
						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
	}

	else if (Typ == OBJ) {
		treal3 pos;
		pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
		treal3 vec;
		vec.x = d_WLLVec.x[i];	vec.y = d_WLLVec.y[i];	vec.z = d_WLLVec.z[i];

		int ix = (int)((d_Pos.x[i] - MINc.x) * DBinv) + 1;
		int iy = (int)((d_Pos.y[i] - MINc.y) * DBinv) + 1;
		int iz = (int)((d_Pos.z[i] - MINc.z) * DBinv) + 1;
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							if (d_Typ[j] == OBJ) {
								treal3 p;//i,j�̋����̐���
								p.x = pos.x - d_Pos.x[j];	p.y = pos.y - d_Pos.y[j];	p.z = pos.z - d_Pos.z[j];
								real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
								if (dist2 < r2) {
									if ((d_WLLVec.x[j] != vec.x) || (d_WLLVec.y[j] != vec.y) || (d_WLLVec.z[j] != vec.z)) {//�߂��̗��q�Ɩ@���x�N�g���Ⴄ�Ȃ�edge
										d_WLLSE[i] = Edge;
									}
								}
							}
						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
	}

	else if (Typ == OBJ2) {
		treal3 pos;
		pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
		treal3 vec;
		vec.x = d_WLLVec.x[i];	vec.y = d_WLLVec.y[i];	vec.z = d_WLLVec.z[i];

		int ix = (int)((d_Pos.x[i] - MINc.x) * DBinv) + 1;
		int iy = (int)((d_Pos.y[i] - MINc.y) * DBinv) + 1;
		int iz = (int)((d_Pos.z[i] - MINc.z) * DBinv) + 1;
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							if (d_Typ[j] == OBJ2) {
								treal3 p;//i,j�̋����̐���
								p.x = pos.x - d_Pos.x[j];	p.y = pos.y - d_Pos.y[j];	p.z = pos.z - d_Pos.z[j];
								real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
								if (dist2 < r2) {
									if ((d_WLLVec.x[j] != vec.x) || (d_WLLVec.y[j] != vec.y) || (d_WLLVec.z[j] != vec.z)) {//�߂��̗��q�Ɩ@���x�N�g���Ⴄ�Ȃ�edge
										d_WLLSE[i] = Edge;
									}
								}
							}
						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
	}

}


void DEMPS::Surface_Edge() {//surface-edge����
	//printf_s("Surface_Edge  start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_Surface_Edge << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_WLLVec, d_WLLSE, WLL_PCL_DST, MINc, DBinv, nBx, nBxy, d_bfst, d_blst, d_nxt);
	//CHECK(hipDeviceSynchronize());

	hipMemcpy(WLLSE, d_WLLSE, sizeof(char) * nP, hipMemcpyDeviceToHost);

	//printf_s("Surface_Edge finished!\n\n");
}


__global__ void d_ResetMRR(const int nP_NumMRR, char* d_TypM, areal3 d_PosM, areal3 d_VelM, real* d_PrsM, int* d_FromWLL)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP_NumMRR) { return; }
	d_TypM[i] = GST;
	d_PosM.x[i] = d_PosM.y[i] = d_PosM.z[i] = 0.0f;
	d_VelM.x[i] = d_VelM.y[i] = d_VelM.z[i] = 0.0f;
	d_PrsM[i] = 0.0f;
	d_FromWLL[i] = -1;
}


void DEMPS::ResetMRR() {//�~���[���q�폜
	//printf_s("ResetMRR start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP * NumMRR);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP_NumMRR(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_ResetMRR << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), d_TypM, d_PosM, d_VelM, d_PrsM, d_FromWLL);
	//CHECK(hipDeviceSynchronize());

	//printf_s("ResetMRR finished!\n\n");
}


__global__ void d_GenMRR_nonslip(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, real* d_Prs, areal3 d_WLLVec, char* d_TypM, areal3 d_PosM, areal3 d_VelM, real* d_PrsM, int* d_FromWLL, char* d_WLLSE, const real r2,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	char Typ = d_Typ[i];
	if (Typ != FLD) { return; }//���̗��q����ʑΏ̃~���[����

	real r2_2 = 1.0f * r2;
	treal3 pos;
	pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
	int WLLexist = 0;//�ߖT�ɕǗ��q�����݂�����P
	int iNM = i * NumMRR;
	int Edgeexist = 0;
	int Edge_unique[NumMRR];
	for (int k = 0; k < NumMRR; k++) { Edge_unique[k] = -1; }

	//�ŋߖT�Ǘ��q�T��
	int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
	int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
	int iz = (int)((pos.z - MINc.z) * DBinv) + 1;
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						char typ = d_Typ[j];
						if ((typ == WLL) || (typ == OBJ) || (typ == OBJ2)) {
							treal3 p;
							p.x = pos.x - d_Pos.x[j];	p.y = pos.y - d_Pos.y[j];	p.z = pos.z - d_Pos.z[j];//�Ǘ��q�����̗��q�̃x�N�g��
							real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
							if (dist2 <= r2_2) {//�e�����a���ɕǗL��
								WLLexist = 1;//�ǗL��t���O
								treal3 wallvec;
								wallvec.x = d_WLLVec.x[j];	wallvec.y = d_WLLVec.y[j];	wallvec.z = d_WLLVec.z[j];

								int unique = 0;//�ʂ̃~���[����
								for (int k = 0; k < NumMRR; k++) {//unique����(i����k�Ԗڂɐ������ꂽ�~���[�Ɩ@���x�N�g�����d�����Ă��Ȃ�������)
									int jNum = d_FromWLL[k + iNM];
									if (jNum != -1) {
										if ((d_WLLVec.x[jNum] == wallvec.x) && (d_WLLVec.y[jNum] == wallvec.y) && (d_WLLVec.z[jNum] == wallvec.z)) {
											unique = 1;//�����@���x�N�g�����o�^�ς�													
										}
									}
								}
								if (unique == 0) {//j��unique�ł���Γo�^
									for (int k = 0; k < NumMRR; k++) {
										int kiNM = k + iNM;
										if (d_FromWLL[kiNM] == -1) {
											d_FromWLL[kiNM] = j;
											break;
										}
									}
								}

								if (d_WLLSE[j] == Edge) {//�p�̃~���[����
									Edgeexist = 1;
									int E_unique = 0;
									for (int k = 0; k < NumMRR; k++) {
										int q = Edge_unique[k];
										if (q != -1) {
											if ((d_WLLVec.x[q] == wallvec.x) && (d_WLLVec.y[q] == wallvec.y) && (d_WLLVec.z[q] == wallvec.z)) {
												E_unique = 1;//�����@���x�N�g�����o�^�ς�														
											}
										}
									}
									if (E_unique == 0) {
										for (int k = 0; k < NumMRR; k++) {
											if (Edge_unique[k] == -1) {
												Edge_unique[k] = j;
												break;
											}
										}
									}
								}

							}
						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	//�~���[���q����
	if (WLLexist == 1) {//�߂��ɕǂ�����

		for (int k = 0; k < NumMRR; k++) {
			int kiNM = k + iNM;
			int FromNum = d_FromWLL[kiNM];//�Ǘ��q�ԍ�j��FromNum�Ƃ��ă��W�X�^�ɓo�^
			if (FromNum == -1) { continue; }//�Ǘ��qj(FromNum)����~���[����	
			treal3 posw;
			posw.x = d_Pos.x[FromNum];	posw.y = d_Pos.y[FromNum];	posw.z = d_Pos.z[FromNum];
			treal3 WLLvec;
			WLLvec.x = d_WLLVec.x[FromNum];	WLLvec.y = d_WLLVec.y[FromNum];	WLLvec.z = d_WLLVec.z[FromNum];
			treal3 PWvec;//�ŋߖT�Ǘ��q�Ɨ��̗��qi�̑��΍��W�x�N�g��
			PWvec.x = pos.x - posw.x;		PWvec.y = pos.y - posw.y;		PWvec.z = pos.z - posw.z;

			real PW_WLL = PWvec.x * WLLvec.x + PWvec.y * WLLvec.y + PWvec.z * WLLvec.z;
			if (PW_WLL < 0) { continue; }//���O����@�����ɂ̓~���[�������Ȃ�(���̂��ǂ̊O�ɂ���)

			real distance = PW_WLL / sqrt(WLLvec.x * WLLvec.x + WLLvec.y * WLLvec.y + WLLvec.z * WLLvec.z);//�����̃X�J���[�l �@���x�N�g���Ȃ̂ŕ���͂P�ɂȂ��Ă�͂�
			d_TypM[kiNM] = MRR;
			d_PosM.x[kiNM] = posw.x + PWvec.x - 2.0f * distance * WLLvec.x;//�@���x�N�g����p���đΏ̈ʒu�ɐ���
			d_PosM.y[kiNM] = posw.y + PWvec.y - 2.0f * distance * WLLvec.y;
			d_PosM.z[kiNM] = posw.z + PWvec.z - 2.0f * distance * WLLvec.z;
			d_VelM.x[kiNM] = -d_Vel.x[i];//non-slip
			d_VelM.y[kiNM] = -d_Vel.y[i];
			d_VelM.z[kiNM] = -d_Vel.z[i];
			d_PrsM[kiNM] = d_Prs[i];
			/*d_VelM.x[kiNM] = 0.0f;
			d_VelM.y[kiNM] = 0.0f;
			d_VelM.z[kiNM] = 0.0f;*/
			//d_PrsM[kiNM] = 0.0f;
		}//�����܂Ŗʃ~���[����

#if 0 //Edge�����t���O
		if (Edgeexist == 1) {
			int NumEdge = 0;
			for (int k = 0; k < NumMRR; k++) {
				if (Edge_unique[k] != -1) { NumEdge++; }
			}

			if (NumEdge >= 2) {//Edge���q���e�����a����2��ވȏ゠�����獇���x�N�g������p�̃~���[����
				treal3 posw;
				treal3 synvec;
				treal3 PWvec;
				real distance;
				for (int k = 0; k < NumMRR; k++) {//�����x�N�g���v�Z+�Ō��edge�𐶐����ɐݒ�	
					int FromNum = Edge_unique[k];
					if (FromNum == -1) { continue; }
					posw.x = d_Pos.x[FromNum];	posw.y = d_Pos.y[FromNum];	posw.z = d_Pos.z[FromNum];
					synvec.x += d_WLLVec.x[FromNum];		synvec.y += d_WLLVec.y[FromNum];		synvec.z += d_WLLVec.z[FromNum];
					PWvec.x = pos.x - posw.x;		PWvec.y = pos.y - posw.y;		PWvec.z = pos.z - posw.z;
				}

				real abs_synvec = sqrt(synvec.x * synvec.x + synvec.y * synvec.y + synvec.z * synvec.z);
				synvec.x /= abs_synvec;	synvec.y /= abs_synvec;	synvec.z /= abs_synvec;//�����x�N�g���̑傫�����P�ɂ���

				real PW_WLL = PWvec.x * synvec.x + PWvec.y * synvec.y + PWvec.z * synvec.z;
				if (PW_WLL < 0) { return; }//���O����@�����ɂ̓~���[�������Ȃ�(���̂��ǂ̊O�ɂ���)

				distance = PW_WLL / sqrt(synvec.x * synvec.x + synvec.y * synvec.y + synvec.z * synvec.z);//�����̃X�J���[�l �@���x�N�g���Ȃ̂ŕ���͂P�ɂȂ��Ă�͂�

				int MRR_space = -1;
				for (int k = 0; k < NumMRR; k++) {
					int kiNM = k + iNM;
					if (d_FromWLL[kiNM] == -1) { MRR_space = kiNM; break; }
				}

				d_TypM[MRR_space] = MRR;
				d_PosM.x[MRR_space] = posw.x + PWvec.x - 2.0f * distance * synvec.x;//�@���x�N�g����p���đΏ̈ʒu�ɐ���
				d_PosM.y[MRR_space] = posw.y + PWvec.y - 2.0f * distance * synvec.y;
				d_PosM.z[MRR_space] = posw.z + PWvec.z - 2.0f * distance * synvec.z;
				/*d_VelM.x[MRR_space] = -d_Vel.x[i];
				d_VelM.y[MRR_space] = -d_Vel.y[i];
				d_VelM.z[MRR_space] = -d_Vel.z[i];
				d_PrsM[MRR_space] = d_Prs[i];*/
				d_VelM.x[MRR_space] = 0.0f;
				d_VelM.y[MRR_space] = 0.0f;
				d_VelM.z[MRR_space] = 0.0f;
				d_PrsM[MRR_space] = d_Prs[i];
			}
		}
#endif
	}

}


void DEMPS::GenMRR_nonslip() {//�~���[�S��������
	//printf_s("GenMRR_nonslip start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP * NumMRR);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nPMRR(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_GenMRR_nonslip << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Prs, d_WLLVec, d_TypM, d_PosM, d_VelM, d_PrsM, d_FromWLL, d_WLLSE, r2, MINc, DBinv, nBx, nBxy, d_bfst, d_blst, d_nxt);
	//CHECK(hipDeviceSynchronize());

	//printf_s("GenMRR_nonslip finished!\n\n");
}


__global__ void d_MkBkt_MRR(const int nP_NumMRR, const  int nBx, const  int nBxy, const  real DBinv, int* d_bfstM, int* d_blstM, int* d_nxtM, const char* d_TypM, const  areal3 d_PosM, const treal3 MINc)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP_NumMRR) { return; }
	if (d_TypM[i] == GST) { return; }
	int ix = (int)((d_PosM.x[i] - MINc.x) * DBinv) + 1;
	int iy = (int)((d_PosM.y[i] - MINc.y) * DBinv) + 1;
	int iz = (int)((d_PosM.z[i] - MINc.z) * DBinv) + 1;
	int ib = iz * nBxy + iy * nBx + ix;
	const int j = atomicExch(&d_blstM[ib], i);
	if (j == -1) { d_bfstM[ib] = i; }
	else { d_nxtM[j] = i; }

}

//�ǂ̓����ɂł����~���[���͂���
__global__ void d_MRRinout(const int nP_NumMRR, const  int nBx, const  int nBxy, const  real DBinv, int* d_bfst, int* d_blst, int* d_nxt, int* d_bfstM, int* d_blstM, int* d_nxtM, char* d_TypM, const  areal3 d_PosM, const char* d_Typ, const areal3 d_Pos, const areal3 d_WLLVec, const real r, const treal3 MINc)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP_NumMRR) { return; }

	treal3 posM;
	posM.x = d_PosM.x[i];	posM.y = d_PosM.y[i];	posM.z = d_PosM.z[i];
	int WLLexist = 0;
	real walldist = 10.0f * r;
	treal3 posw = { 0.0f };
	treal3 wallvec = { 0.0f };

	int ix = (int)((posM.x - MINc.x) * DBinv) + 1;
	int iy = (int)((posM.y - MINc.y) * DBinv) + 1;
	int iz = (int)((posM.z - MINc.z) * DBinv) + 1;

	//�ŋߖT�Ǘ��q�T��
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n

					char typ = d_Typ[j];
					if ((typ == WLL) || (typ == OBJ) || (typ == OBJ2)) {
						treal3 posj;
						posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];
						treal3 p;
						p.x = posM.x - posj.x;	p.y = posM.y - posj.y;	p.z = posM.z - posj.z;
						real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
						real dist = sqrt(dist2);
						if (dist <= r) {//�e�����a���ɕǗL��
							if (dist < walldist) {
								WLLexist = 1;//�ǗL��t���O j�ɂ���
								walldist = dist;
								wallvec.x = d_WLLVec.x[j];	wallvec.y = d_WLLVec.y[j];	wallvec.z = d_WLLVec.z[j];
								posw.x = posj.x;	posw.y = posj.y;	posw.z = posj.z;
							}
						}
					}

					j = d_nxt[j];
					if (j == -1) break;
				}
			}
		}
	}

	real inout = (posM.x - posw.x) * wallvec.x + (posM.y - posw.y) * wallvec.y + (posM.z - posw.z) * wallvec.z;//�Ǘ��q���~���[���q�̃x�N�g�� * �ǖ@���x�N�g��
	if ((WLLexist == 1) && (inout >= 0)) { d_TypM[i] = GST; }//���O����Ɉ�������������GST��

}

void DEMPS::MkBkt_MRR() {//���q���o�P�b�g�Ɏ��[
	//printf_s("MkBkt_MRR start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP * NumMRR);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP_NumMRR(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_bfstM, -1)));
	((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_blstM, -1)));
	((d_initialize_int_array << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), d_nxtM, -1)));
	d_MkBkt_MRR << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), nBx, nBxy, DBinv, d_bfstM, d_blstM, d_nxtM, d_TypM, d_PosM, MINc);

	d_MRRinout << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), nBx, nBxy, DBinv, d_bfst, d_blst, d_nxt, d_bfstM, d_blstM, d_nxtM, d_TypM, d_PosM, d_Typ, d_Pos, d_WLLVec, r, MINc);

	/*((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_bfstM, -1)));//inout��GST�ɂȂ�����̓o�P�b�g�ɓ���Ȃ��@�����@�Ăяo������GST�Ȃ�continue�̏������ꂽ
	((d_initialize_int_array << <blocks_nBxyz, threads >> > (nBxyz, d_blstM, -1)));
	((d_initialize_int_array << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), d_nxtM, -1)));
	d_MkBkt_MRR << <blocks_nP_NumMRR, threads >> > ((nP * NumMRR), nBx, nBxy, DBinv, d_bfstM, d_blstM, d_nxtM, d_TypM, d_PosM, MINc);*/
	//CHECK(hipDeviceSynchronize());

	//printf_s("MkBkt_MRR finished!\n\n");
}


__global__ void d_VscTrm(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, areal3 d_WLLVec, char* d_TypM, areal3 d_PosM, areal3 d_VelM, const real r, const real PCL_DST, const real n0, const real KNM_VSC, const real Vsc_coef, treal3 G,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_nxt, const int* d_bfstM, const int* d_nxtM)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	if (d_Typ[i] != FLD) { return; }

	treal3 pos;		treal3 vel;
	pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
	vel.x = d_Vel.x[i];		vel.y = d_Vel.y[i];		vel.z = d_Vel.z[i];
	treal3 Acc;
	Acc.x = Acc.y = Acc.z = 0.0f;//�����x�̈ꎞ�v�Z
	real invn0 = 1.0f / n0;

	int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
	int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
	int iz = (int)((pos.z - MINc.z) * DBinv) + 1;

	//���̃��[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						if (d_Typ[j] == FLD) {
							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2;// i,j������2��
							dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
							real dist;//���q�̋���(��Βl)
							dist = sqrt(dist2);
							if (dist <= r) {
								/*real w = WEI(dist, r);//i,j�̏d��
								Acc.x += (d_Vel.x[j] - vel.x) * w;
								Acc.y += (d_Vel.y[j] - vel.y) * w;
								Acc.z += (d_Vel.z[j] - vel.z) * w;*/
								Acc.x += 2.0f * (d_Vel.x[j] - vel.x) * PCL_DST / dist / dist / dist;
								Acc.y += 2.0f * (d_Vel.y[j] - vel.y) * PCL_DST / dist / dist / dist;
								Acc.z += 2.0f * (d_Vel.z[j] - vel.z) * PCL_DST / dist / dist / dist;
							}

						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}
	//�~���[���[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfstM[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					//���O����
					treal3 posMj;
					posMj.x = d_PosM.x[j];	posMj.y = d_PosM.y[j];	posMj.z = d_PosM.z[j];


					treal3 p;//i,j�̋����̐���
					p.x = posMj.x - pos.x;	p.y = posMj.y - pos.y;	p.z = posMj.z - pos.z;
					real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
					real dist = sqrt(dist2);//���q�̋���(��Βl)
					if (dist <= r) {
						/*real w = WEI(dist, r);//i,j�̏d��
						Acc.x += (d_VelM.x[j] - vel.x) * w;
						Acc.y += (d_VelM.y[j] - vel.y) * w;
						Acc.z += (d_VelM.z[j] - vel.z) * w;*/
						Acc.x += 2.0f * (d_VelM.x[j] - vel.x) * PCL_DST / dist / dist / dist;
						Acc.y += 2.0f * (d_VelM.y[j] - vel.y) * PCL_DST / dist / dist / dist;
						Acc.z += 2.0f * (d_VelM.z[j] - vel.z) * PCL_DST / dist / dist / dist;
					}

					j = d_nxtM[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	real n0KNM = invn0 * KNM_VSC;
	d_Acc.x[i] = n0KNM * Acc.x + G.x;
	d_Acc.y[i] = n0KNM * Acc.y + G.y;
	d_Acc.z[i] = n0KNM * Acc.z + G.z;

}


void DEMPS::VscTrm() {//�S�����E�O�͍��̌v�Z
	//printf_s("VscTrm start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_VscTrm << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Acc, d_WLLVec, d_TypM, d_PosM, d_VelM, r, PCL_DST, n0, KNM_VSC, Vsc_coef, G, MINc, DBinv, nBx, nBxy, d_bfst, d_nxt, d_bfstM, d_nxtM);
	//CHECK(hipDeviceSynchronize());

	//printf_s("VscTrm finished!\n\n");
}


__device__ void d_ChkPcl(const int i, char* d_Typ, areal3 d_Pos, areal3 d_Vel, const  areal3 d_Acc, real* d_Prs, const  treal3 MINc, const  treal3 MAXc, const real PCL_DST, const real ulmax)//&�v��H
{
	//if (d_Typ[i] != FLD) { return; }//�Ăяo�����ł��̏������Ă�@�Ǔ������Ƃ��͕ύX���邩��

	if (d_Pos.x[i] < (MAXc.x - 3.1f * PCL_DST) && d_Pos.x[i] > (MINc.x + 3.1f * PCL_DST) &&
		d_Pos.y[i] < (MAXc.y - 3.1f * PCL_DST) && d_Pos.y[i] > (MINc.y + 3.1f * PCL_DST) &&
		d_Pos.z[i] < (MAXc.z - 3.1f * PCL_DST) && d_Pos.z[i] > (MINc.z + 3.1f * PCL_DST)) {//�ő呬�x�ɐ���

		treal3 Utmp;
		Utmp.x = d_Vel.x[i];	Utmp.y = d_Vel.y[i];	Utmp.z = d_Vel.z[i];
		real U = Utmp.x * Utmp.x + Utmp.y * Utmp.y + Utmp.z * Utmp.z;
		U = sqrt(U);
		if (U > ulmax) {
			Utmp.x *= ulmax / U;	Utmp.y *= ulmax / U;	Utmp.z *= ulmax / U;
			d_Vel.x[i] = Utmp.x;		d_Vel.y[i] = Utmp.y;		d_Vel.z[i] = Utmp.z;
		}

	}
	else {//���U���Ă��闱�q�͍폜
		d_Typ[i] = GST;
		d_Pos.x[i] = d_Pos.y[i] = d_Pos.z[i] = 0.0f;
		d_Vel.x[i] = d_Vel.y[i] = d_Vel.z[i] = 0.0f;
		d_Acc.x[i] = d_Acc.y[i] = d_Acc.z[i] = 0.0f;
		d_Prs[i] = 0.0f;
	}

}


__global__ void d_UpPcl1(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, real* d_Prs, const  treal3 MINc, const  treal3 MAXc, const  real dt, const real  PCL_DST, const real ulmax)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	if (d_Typ[i] != FLD) { return; }

	d_Vel.x[i] += d_Acc.x[i] * dt;	d_Vel.y[i] += d_Acc.y[i] * dt;	d_Vel.z[i] += d_Acc.z[i] * dt;
	d_Pos.x[i] += d_Vel.x[i] * dt;	d_Pos.y[i] += d_Vel.y[i] * dt;	d_Pos.z[i] += d_Vel.z[i] * dt;
	d_Acc.x[i] = d_Acc.y[i] = d_Acc.z[i] = 0.0f;

	d_ChkPcl(i, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs, MINc, MAXc, PCL_DST, ulmax);

}


void DEMPS::UpPcl1() {//���̗��q�ړ�
	//printf_s("UpPcl1 start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_UpPcl1 << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs, MINc, MAXc, dt, PCL_DST, ulmax);
	//CHECK(hipDeviceSynchronize());

	//printf_s("UpPcl1 finished!\n\n");
}


__global__ void d_ChkCol(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, areal3 d_WLLVec, char* d_TypM, areal3 d_PosM, areal3 d_VelM, real* d_Dns, const real PCL_DST, const real r, const real rlim2, const real COL,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_nxt, const int* d_bfstM, const int* d_nxtM)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	char Typ = d_Typ[i];
	if (Typ != FLD) { return; }

	treal3 pos;		treal3 vec;	treal3 vec2;
	pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
	vec.x = d_Vel.x[i];		vec.y = d_Vel.y[i];		vec.z = d_Vel.z[i];
	vec2.x = d_Vel.x[i];		vec2.y = d_Vel.y[i];		vec2.z = d_Vel.z[i];
	real mi = d_Dns[Typ];
	real rlim_wall = 0.45f * PCL_DST;
	real rlim_wall2 = rlim_wall * rlim_wall;

	int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
	int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
	int iz = (int)((pos.z - MINc.z) * DBinv) + 1;

	//���̃��[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						char typ = d_Typ[j];
						if (typ == FLD) {
							//���O����
							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
							if (dist2 < rlim2) {
								real fDT = (vec.x - d_Vel.x[j]) * p.x + (vec.y - d_Vel.y[j]) * p.y + (vec.z - d_Vel.z[j]) * p.z;
								if (fDT > 0.0f) {
									real mj = d_Dns[typ];
									fDT *= COL * mj / (mi + mj) / dist2;
									vec2.x -= p.x * fDT;	vec2.y -= p.y * fDT;	vec2.z -= p.z * fDT;
								}
							}

						}
						else if ((typ == WLL) || (typ == OBJ) || (typ == OBJ2)) {

							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];
							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
							if (dist2 < rlim_wall2) {
								real fDT = (vec.x - d_Vel.x[j]) * p.x + (vec.y - d_Vel.y[j]) * p.y + (vec.z - d_Vel.z[j]) * p.z;
								if (fDT > 0.0f) {
									real mj = d_Dns[typ];
									fDT *= COL * mj / (mi + mj) / dist2;
									vec2.x -= p.x * fDT;	vec2.y -= p.y * fDT;	vec2.z -= p.z * fDT;
								}
							}
						}

					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}
	//�~���[���[�v
	real mj = d_Dns[MRR];
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfstM[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					//���O����
					treal3 posMj;
					posMj.x = d_PosM.x[j];	posMj.y = d_PosM.y[j];	posMj.z = d_PosM.z[j];

					treal3 p;//i,j�̋����̐���
					p.x = posMj.x - pos.x;	p.y = posMj.y - pos.y;	p.z = posMj.z - pos.z;
					real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
					if (dist2 < rlim2) {
						real fDT = (vec.x - d_VelM.x[j]) * p.x + (vec.y - d_VelM.y[j]) * p.y + (vec.z - d_VelM.z[j]) * p.z;
						if (fDT > 0.0f) {
							fDT *= COL * mj / (mi + mj) / dist2;
							vec2.x -= p.x * fDT;	vec2.y -= p.y * fDT;	vec2.z -= p.z * fDT;
						}
					}

					j = d_nxtM[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}
	d_Acc.x[i] = vec2.x;	d_Acc.y[i] = vec2.y;	d_Acc.z[i] = vec2.z;

}


void __global__ d_UpChkCol(const int nP, char* d_Typ, areal3 d_Vel, areal3 d_Acc) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	if (d_Typ[i] != FLD) { return; }
	d_Vel.x[i] = d_Acc.x[i];		d_Vel.y[i] = d_Acc.y[i];		d_Vel.z[i] = d_Acc.z[i];
}

void DEMPS::ChkCol() {//���̗��q�ړ�
	//printf_s("ChkCol start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_ChkCol << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Acc, d_WLLVec, d_TypM, d_PosM, d_VelM, d_Dns, PCL_DST, r, rlim2, COL, MINc, DBinv, nBx, nBxy, d_bfst, d_nxt, d_bfstM, d_nxtM);
	d_UpChkCol << <blocks_nP, threads >> > (nP, d_Typ, d_Vel, d_Acc);
	//CHECK(hipDeviceSynchronize());

	//printf_s("ChkCol finished!\n\n");
}


__global__ void d_MkPrs(const int nP, char* d_Typ, areal3 d_Pos, real* d_Dns, real* d_Prs, areal3 d_WLLVec, char* d_TypM, areal3 d_PosM, const real rp, const real n0_grad, const real Pmax, const real Prs_coef,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_nxt, const int* d_bfstM, const int* d_nxtM)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	char Typ = d_Typ[i];
	if (Typ != FLD) { return; }

	treal3 pos;
	pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
	real ni = 0.0f;

	int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
	int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
	int iz = (int)((pos.z - MINc.z) * DBinv) + 1;



	//���̃��[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						if (d_Typ[j] == FLD) {
							//���O����
							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
							real dist = sqrt(dist2);;
							if (dist < rp) {
								ni += WEI_grad(dist, rp);
							}

						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	//�~���[���[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfstM[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					//���O����
					treal3 posMj;
					posMj.x = d_PosM.x[j];	posMj.y = d_PosM.y[j];	posMj.z = d_PosM.z[j];

					treal3 p;//i,j�̋����̐���
					p.x = posMj.x - pos.x;	p.y = posMj.y - pos.y;	p.z = posMj.z - pos.z;
					real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
					real dist = sqrt(dist2);
					if (dist < rp) {
						ni += WEI_grad(dist, rp);
					}

					j = d_nxtM[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	real mi = d_Dns[Typ];
	real pressure = (ni > n0_grad) * (ni - n0_grad) * Prs_coef * mi;

	if (pressure > Pmax)
	{
		pressure = Pmax;//�ő�l�}��
	}
	else if (pressure < 0.0f)
	{
		pressure = 0.0f;//������0��
	}

	d_Prs[i] = pressure;

}


void DEMPS::MkPrs() {//���̗��q�ړ�
	//printf_s("MkPrs start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_MkPrs << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Dns, d_Prs, d_WLLVec, d_TypM, d_PosM, rp, n0_grad, Pmax, Prs_coef, MINc, DBinv, nBx, nBxy, d_bfst, d_nxt, d_bfstM, d_nxtM);
	//CHECK(hipDeviceSynchronize());

	//printf_s("MkPrs finished!\n\n");
}


#if 0
__global__ void d_PrsGrdTrm(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Acc, real* d_Prs, areal3 d_WLLVec, real* d_Dns, char* d_TypM, areal3 d_PosM, real* d_PrsM, const real rp, const real rp2, const real n0_grad,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_nxt, const int* d_bfstM, const int* d_nxtM)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < nP) {
		char Typ = d_Typ[i];
		if (Typ == FLD) {
			real invn0 = 1.0f / n0_grad;
			real invro = -3.0f / d_Dns[Typ];//-DIM/�όW��
			treal3 pos;
			pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
			real Prs_min = d_Prs[i];
			/*real invLeft[9] = {0.0f};//���͌��z�����u���b�N�̃C���o�[�X����O��3*3�e���\��
			real Left[9] = { 0.0f };//���͌��z���u���b�N��3*3�e���\��
			real Right[3] = { 0.0f };//���͌��z�E�u���b�N�̈��͂����Ă�x�N�g��*/

			real invLeft0 = 0.0f;
			real invLeft1 = 0.0f;
			real invLeft2 = 0.0f;
			real invLeft3 = 0.0f;
			real invLeft4 = 0.0f;
			real invLeft5 = 0.0f;
			real invLeft6 = 0.0f;
			real invLeft7 = 0.0f;
			real invLeft8 = 0.0f;

			real Left0 = 0.0f;
			real Left1 = 0.0f;
			real Left2 = 0.0f;
			real Left3 = 0.0f;
			real Left4 = 0.0f;
			real Left5 = 0.0f;
			real Left6 = 0.0f;
			real Left7 = 0.0f;
			real Left8 = 0.0f;

			real Right0 = 0.0f;
			real Right1 = 0.0f;
			real Right2 = 0.0f;

			int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
			int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
			int iz = (int)((pos.z - MINc.z) * DBinv) + 1;


			//�ߖT�ŏ����͒��o
			//���̃��[�v
			for (int jz = iz - 1; jz <= iz + 1; jz++) {
				for (int jy = iy - 1; jy <= iy + 1; jy++) {
					for (int jx = ix - 1; jx <= ix + 1; jx++) {
						int jb = jz * nBxy + jy * nBx + jx;
						int j = d_bfst[jb];
						if (j == -1) continue;
						for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
							if (j != i) {
								if (d_Typ[j] == FLD) {
									//���O����
									treal3 posj;
									posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

									treal3 p;//i,j�̋����̐���
									p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
									real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
									if (dist2 < rp2) {
										real prs = d_Prs[j];
										if (prs < Prs_min) {
											Prs_min = prs;
										}
									}

								}
							}
							j = d_nxt[j];
							if (j == -1) break;
						}//���qi�̋ߖT���qj�̃��[�v�I��
					}
				}
			}


			//���͍��v�Z  �����E�������f��
			//���̃��[�v
			for (int jz = iz - 1; jz <= iz + 1; jz++) {
				for (int jy = iy - 1; jy <= iy + 1; jy++) {
					for (int jx = ix - 1; jx <= ix + 1; jx++) {
						int jb = jz * nBxy + jy * nBx + jx;
						int j = d_bfst[jb];
						if (j == -1) continue;
						for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
							if (j != i) {
								if (d_Typ[j] == FLD) {
									//���O����
									treal3 posj;
									posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

									treal3 p;//i,j�̋����̐���
									p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
									real dist2;// i,j������2��
									dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
									if (dist2 < rp2) {
										real dist = sqrt(dist2);
										real w = WEI_grad(dist, rp) / dist2;
										real Prsj_min = d_Prs[j] - Prs_min;
										//real Prsj_min = d_Prs[j] + d_Prs[i];
										invLeft0 += w * p.x * p.x;
										invLeft1 += w * p.x * p.y;
										invLeft2 += w * p.x * p.z;
										invLeft3 += w * p.y * p.x;
										invLeft4 += w * p.y * p.y;
										invLeft5 += w * p.y * p.z;
										invLeft6 += w * p.z * p.x;
										invLeft7 += w * p.z * p.y;
										invLeft8 += w * p.z * p.z;
										Right0 += w * Prsj_min * p.x;
										Right1 += w * Prsj_min * p.y;
										Right2 += w * Prsj_min * p.z;
									}

								}
							}
							j = d_nxt[j];
							if (j == -1) break;
						}//���qi�̋ߖT���qj�̃��[�v�I��
					}
				}
			}
			//�~���[���[�v
			for (int jz = iz - 1; jz <= iz + 1; jz++) {
				for (int jy = iy - 1; jy <= iy + 1; jy++) {
					for (int jx = ix - 1; jx <= ix + 1; jx++) {
						int jb = jz * nBxy + jy * nBx + jx;
						int j = d_bfstM[jb];
						if (j == -1) continue;
						for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
							if (j != i) {
								//���O����
								treal3 posMj;
								posMj.x = d_PosM.x[j];	posMj.y = d_PosM.y[j];	posMj.z = d_PosM.z[j];


								treal3 p;//i,j�̋����̐���
								p.x = posMj.x - pos.x;	p.y = posMj.y - pos.y;	p.z = posMj.z - pos.z;
								real dist2;// i,j������2��
								dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
								if (dist2 < rp2) {
									real dist = sqrt(dist2);
									real w = WEI_grad(dist, rp) / dist2;
									real PrsMj_min = d_PrsM[j] - Prs_min;
									//real PrsMj_min = d_PrsM[j] + d_Prs[i];
									invLeft0 += w * p.x * p.x;
									invLeft1 += w * p.x * p.y;
									invLeft2 += w * p.x * p.z;
									invLeft3 += w * p.y * p.x;
									invLeft4 += w * p.y * p.y;
									invLeft5 += w * p.y * p.z;
									invLeft6 += w * p.z * p.x;
									invLeft7 += w * p.z * p.y;
									invLeft8 += w * p.z * p.z;
									Right0 += w * PrsMj_min * p.x;
									Right1 += w * PrsMj_min * p.y;
									Right2 += w * PrsMj_min * p.z;
								}

							}
							j = d_nxtM[j];
							if (j == -1) break;
						}//���qi�̋ߖT���qj�̃��[�v�I��
					}
				}
			}


			invLeft0 = invn0 * invLeft0;//n0�Ŋ���
			invLeft1 = invn0 * invLeft1;
			invLeft2 = invn0 * invLeft2;
			invLeft3 = invn0 * invLeft3;
			invLeft4 = invn0 * invLeft4;
			invLeft5 = invn0 * invLeft5;
			invLeft6 = invn0 * invLeft6;
			invLeft7 = invn0 * invLeft7;
			invLeft8 = invn0 * invLeft8;

			//if ((invLeft0 == 0.0f) && (invLeft1 == 0.0f) && (invLeft2 == 0.0f) && (invLeft3 == 0.0f) && (invLeft4 == 0.0f) && (invLeft5 == 0.0f) && (invLeft6 == 0.0f) && (invLeft7 == 0.0f) && (invLeft8 == 0.0f)) { return; }//�f�B�^�[�~�i���g�v�Z��0�����Ȃ��悤��

			real DET_Left = 1.0f / ((invLeft0 * invLeft4 * invLeft8 + invLeft1 * invLeft5 * invLeft6 + invLeft2 * invLeft3 * invLeft7) - (invLeft2 * invLeft4 * invLeft6 + invLeft0 * invLeft5 * invLeft7 + invLeft1 * invLeft3 * invLeft8));//��������C���o�[�X�v�Z
			if (DET_Left == 0.0f) { return; }//�f�B�^�[�~�i���g�v�Z��0�����Ȃ��悤��
			Left0 = DET_Left * (invLeft4 * invLeft8 - invLeft5 * invLeft7);
			Left1 = DET_Left * -(invLeft1 * invLeft8 - invLeft2 * invLeft7);
			Left2 = DET_Left * (invLeft1 * invLeft5 - invLeft2 * invLeft4);
			Left3 = DET_Left * -(invLeft3 * invLeft8 - invLeft5 * invLeft6);
			Left4 = DET_Left * (invLeft0 * invLeft8 - invLeft2 * invLeft6);
			Left5 = DET_Left * -(invLeft0 * invLeft5 - invLeft2 * invLeft3);
			Left6 = DET_Left * (invLeft3 * invLeft7 - invLeft4 * invLeft6);
			Left7 = DET_Left * -(invLeft0 * invLeft7 - invLeft1 * invLeft6);
			Left8 = DET_Left * (invLeft0 * invLeft4 - invLeft1 * invLeft3);

			Right0 = invn0 * Right0;//n0�Ŋ���
			Right1 = invn0 * Right1;
			Right2 = invn0 * Right2;



			d_Acc.x[i] = invro * (Left0 * Right0 + Left1 * Right1 + Left2 * Right2);
			d_Acc.y[i] = invro * (Left3 * Right0 + Left4 * Right1 + Left5 * Right2);
			d_Acc.z[i] = invro * (Left6 * Right0 + Left7 * Right1 + Left8 * Right2);


		}
	}
}

#else
__global__ void d_PrsGrdTrm(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Acc, real* d_Prs, areal3 d_WLLVec, real* d_Dns, char* d_TypM, areal3 d_PosM, real* d_PrsM, const real rp, const real rp2, const real n0_grad,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_nxt, const int* d_bfstM, const int* d_nxtM)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	char Typ = d_Typ[i];
	if (Typ != FLD) { return; }

	//real invn0 = 1.0f / n0_grad;
	real invro = -3.0f / d_Dns[Typ];//-DIM/�όW��
	treal3 pos;
	pos.x = d_Pos.x[i];	pos.y = d_Pos.y[i];	pos.z = d_Pos.z[i];
	real Prs_min = d_Prs[i];

	treal3 Acc = { 0.0f };
	real A3 = 3.0f / n0_grad;//Dimension / n0

	int ix = (int)((pos.x - MINc.x) * DBinv) + 1;
	int iy = (int)((pos.y - MINc.y) * DBinv) + 1;
	int iz = (int)((pos.z - MINc.z) * DBinv) + 1;



	//�ߖT�ŏ����͒��o
	//���̃��[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						if (d_Typ[j] == FLD) {
							//���O����
							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2 = p.x * p.x + p.y * p.y + p.z * p.z;// i,j������2��
							if (dist2 < rp2) {
								real prs = d_Prs[j];
								if (prs < Prs_min) {
									Prs_min = prs;
								}
							}

						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	//���̃��[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						if (d_Typ[j] == FLD) {

							treal3 posj;
							posj.x = d_Pos.x[j];	posj.y = d_Pos.y[j];	posj.z = d_Pos.z[j];

							treal3 p;//i,j�̋����̐���
							p.x = posj.x - pos.x;	p.y = posj.y - pos.y;	p.z = posj.z - pos.z;
							real dist2;// i,j������2��
							dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
							if (dist2 < rp2) {
								real dist = sqrt(dist2);
								real w = WEI_grad(dist, rp) / dist2;
								real Prsj_min = d_Prs[j] - Prs_min;
								Acc.x += Prsj_min * w * p.x;
								Acc.y += Prsj_min * w * p.y;
								Acc.z += Prsj_min * w * p.z;
								/*real Prs = d_Prs[j] +d_Prs[i];
								Acc.x += Prs * w * p.x;
								Acc.y += Prs * w * p.y;
								Acc.z += Prs * w * p.z;*/
							}

						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}
	//�~���[���[�v
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfstM[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						//���O����
						treal3 posMj;
						posMj.x = d_PosM.x[j];	posMj.y = d_PosM.y[j];	posMj.z = d_PosM.z[j];

						treal3 p;//i,j�̋����̐���
						p.x = posMj.x - pos.x;	p.y = posMj.y - pos.y;	p.z = posMj.z - pos.z;
						real dist2;// i,j������2��
						dist2 = p.x * p.x + p.y * p.y + p.z * p.z;
						if (dist2 < rp2) {
							real dist = sqrt(dist2);
							real w = WEI_grad(dist, rp) / dist2;
							real PrsMj_min = d_PrsM[j] - Prs_min;
							Acc.x += PrsMj_min * w * p.x;
							Acc.y += PrsMj_min * w * p.y;
							Acc.z += PrsMj_min * w * p.z;
							/*real Prs = d_PrsM[j] + d_Prs[i];
							Acc.x += Prs * w * p.x;
							Acc.y += Prs * w * p.y;
							Acc.z += Prs * w * p.z;*/
						}

					}
					j = d_nxtM[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

	d_Acc.x[i] = invro * Acc.x * A3;
	d_Acc.y[i] = invro * Acc.y * A3;
	d_Acc.z[i] = invro * Acc.z * A3;

}
#endif


void DEMPS::PrsGrdTrm() {//���̗��q�ړ�

	//printf_s("PrsGrdTrm start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_PrsGrdTrm << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Acc, d_Prs, d_WLLVec, d_Dns, d_TypM, d_PosM, d_PrsM, rp, rp2, n0_grad, MINc, DBinv, nBx, nBxy, d_bfst, d_nxt, d_bfstM, d_nxtM);
	//CHECK(hipDeviceSynchronize());

	//printf_s("PrsGrdTrm finished!\n\n");
}


__global__ void d_UpPcl2(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, real* d_Prs, real* d_pav, const  treal3 MINc, const  treal3 MAXc, const  real dt, const real  PCL_DST, const real ulmax)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	if (d_Typ[i] != FLD) { return; }

	treal3 acc;
	acc.x = d_Acc.x[i];	acc.y = d_Acc.y[i];	acc.z = d_Acc.z[i];
	d_Vel.x[i] += acc.x * dt;	d_Vel.y[i] += acc.y * dt;	d_Vel.z[i] += acc.z * dt;
	d_Pos.x[i] += acc.x * dt * dt;	d_Pos.y[i] += acc.y * dt * dt;	d_Pos.z[i] += acc.z * dt * dt;
	d_Acc.x[i] = d_Acc.y[i] = d_Acc.z[i] = 0.0f;

	d_pav[i] += d_Prs[i];//���ԕ��ψ��͉��Z

	d_ChkPcl(i, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs, MINc, MAXc, PCL_DST, ulmax);

}

void DEMPS::UpPcl2() {//���͏C�����̗��q�ړ�
	//printf_s("UpPcl2 start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_UpPcl2 << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Acc, d_Prs, d_pav, MINc, MAXc, dt, PCL_DST, ulmax);
	//CHECK(hipDeviceSynchronize());

	//printf_s("UpPcl2 finished!\n\n");
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////MPS_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif


#if DEM_flg
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////DEM_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int sign(real a, real b) {//a�̐�Βl��b�̕���������
	int c = (b >= 0.0f) - (b <= 0.0f);
	return abs(a) * c;
}

__device__ real PairNumber(int* pair, int i, int j) {//���qi�̐ڐG�����ƍ�
	int contact;
	int count = 0;

	for (int p = 0; p < NCP; p++) {//�ڐG��������@�ꏊ����
		if (pair[p + i * NCP] == j) {
			contact = p;
			break;
		}
		else { count += 1; }
	}

	if (count == NCP) {//�ڐG�����Ȃ��@�V�K�o�^
		for (int q = 0; q < NCP; q++) {
			if (pair[q + i * NCP] == -2) {
				pair[q + i * NCP] = j;
				contact = q;
				break;
			}
		}
	}
	return contact;
}


__global__ void d_ColForce(const int nP, const int nPSLD, const int nPWLL, const real* d_D, const char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque,
	areal3 d_ep, int* d_pair, const real m, const real eta_n, const real eta_t, real kn, real kt, const real mu, const real dt,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	if (d_Typ[i] != SLD) { return; }

	real Di = d_D[i];
	treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
	treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
	treal3 Omegai;	Omegai.x = d_Omega.x[i];	Omegai.y = d_Omega.y[i];	Omegai.z = d_Omega.z[i];

	int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
	int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
	int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;
	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						if (d_Typ[j] != FLD) {
							treal3 r_delta_i;
							treal3 r_delta_j;
							treal3 Xi;

							treal3 dist;
							real L;
							real L_2;
							treal3 n;

							treal3 Fcol;
							real Tr = 0.0f;
							treal3 dp;

							real Dj = d_D[j];
							treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
							treal3 Velj;		Velj.x = d_Vel.x[j];		Velj.y = d_Vel.y[j];		Velj.z = d_Vel.z[j];
							treal3 Omegaj;	Omegaj.x = d_Omega.x[j];	Omegaj.y = d_Omega.y[j];	Omegaj.z = d_Omega.z[j];

							dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;

							L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);

							n.x = dist.x / L;//l   
							n.y = dist.y / L;//m
							n.z = dist.z / L;//n

							L_2 = sqrt(n.x * n.x + n.y * n.y);//�����]���̒��g

							if (L - 0.5f * (Di + Dj) < 0.0f) {//�ڐG����

								int couple = PairNumber(d_pair, i, j) + i * NCP; //���qi,j��pair�z��ԍ� (i - nPWLL)����ύX�@�O�͕Ǘ��q���擪����������
								treal3 ep;	ep.x = d_ep.x[couple];		ep.y = d_ep.y[couple];		ep.z = d_ep.z[couple];

								if (L_2 == 0) {//�W���o�����b�N�΍�

									r_delta_i.x = -Omegai.z * dt;
									r_delta_i.y = Omegai.y * dt;
									r_delta_i.z = Omegai.x * dt;
									r_delta_j.x = -Omegaj.z * dt;
									r_delta_j.y = Omegaj.y * dt;
									r_delta_j.z = Omegaj.x * dt;//�p�ψ�

									Xi.x = -(Veli.z - Velj.z) * dt;
									Xi.y = (Veli.y - Velj.y) * dt + (r_delta_i.z * Di + r_delta_j.z * Dj) * 0.5f;
									Xi.z = (Veli.x - Velj.x) * dt - (r_delta_i.y * Di + r_delta_j.y * Dj) * 0.5f;

									//////////////////////���[�J��x///////////////////////////////
									ep.x += kn * Xi.x;
									dp.x = eta_n * Xi.x / dt;
									if (ep.x < 0.0f) { ep.x = dp.x = 0.0f; }
									d_ep.x[couple] = ep.x;//�o�l�X�V
									Fcol.x = ep.x + dp.x;
									//////////////////////���[�J��x///////////////////////////////

									//////////////////////���[�J��y///////////////////////////////
									ep.y += kt * Xi.y;
									dp.y = eta_t * Xi.y / dt;
									if (ep.x < 0.0f) { ep.y = dp.y = 0.0f; }
									if (abs(ep.y) > mu * ep.x) { ep.y = mu * sign(ep.x, ep.y);		dp.y = 0.0f; }
									d_ep.y[couple] = ep.y;
									Fcol.y = ep.y + dp.y;
									//////////////////////���[�J��y///////////////////////////////

									//////////////////////���[�J��z///////////////////////////////
									ep.z += kt * Xi.z;
									dp.z = eta_t * Xi.z / dt;
									if (ep.x < 0.0f) { ep.z = dp.z = 0.0f; }
									if (abs(ep.z) > mu * ep.x) { ep.z = mu * sign(ep.x, ep.z);		dp.z = 0.0f; }
									d_ep.z[couple] = ep.z;
									Fcol.z = ep.z + dp.z;
									//////////////////////���[�J��z///////////////////////////////


									//////////////////////���[���h///////////////////////////////
									d_Ftotal.x[i] -= Fcol.z;
									d_Ftotal.y[i] -= Fcol.y;
									d_Ftotal.z[i] -= -Fcol.x;

									d_Torque.x[i] -= Fcol.y * Di * 0.5f;
									d_Torque.y[i] -= -Fcol.z * Di * 0.5f;
									d_Torque.z[i] -= -Tr;
									//////////////////////���[���h///////////////////////////////

								}
								else {//�ʏ�

									r_delta_i.x = (n.x * Omegai.x + n.y * Omegai.y + n.z * Omegai.z) * dt;
									r_delta_i.y = (-n.y * Omegai.x / L_2 + n.x * Omegai.y / L_2) * dt;
									r_delta_i.z = (-n.x * n.z * Omegai.x / L_2 - n.y * n.z * Omegai.y / L_2 + L_2 * Omegai.z) * dt;
									r_delta_j.x = (n.x * Omegaj.x + n.y * Omegaj.y + n.z * Omegaj.z) * dt;
									r_delta_j.y = (-n.y * Omegaj.x / L_2 + n.x * Omegaj.y / L_2) * dt;
									r_delta_j.z = (-n.x * n.z * Omegaj.x / L_2 - n.y * n.z * Omegaj.y / L_2 + L_2 * Omegaj.z) * dt;//�p�ψ�

									Xi.x = (n.x * (Veli.x - Velj.x) + n.y * (Veli.y - Velj.y) + n.z * (Veli.z - Velj.z)) * dt;
									Xi.y = (-n.y * (Veli.x - Velj.x) / L_2 + n.x * (Veli.y - Velj.y) / L_2) * dt + (r_delta_i.z * Di + r_delta_j.z * Dj) * 0.5f;
									Xi.z = (-n.x * n.z * (Veli.x - Velj.x) / L_2 - n.y * n.z * (Veli.y - Velj.y) / L_2 + (Veli.z - Velj.z) * L_2) * dt - (r_delta_i.y * Di + r_delta_j.y * Dj) * 0.5f;

									//////////////////////���[�J��x///////////////////////////////
									ep.x += kn * Xi.x;
									dp.x = eta_n * Xi.x / dt;
									if (ep.x < 0.0f) { ep.x = dp.x = 0.0f; }
									d_ep.x[couple] = ep.x;//�o�l�X�V
									Fcol.x = ep.x + dp.x;
									//////////////////////���[�J��x///////////////////////////////

									//////////////////////���[�J��y///////////////////////////////
									ep.y += kt * Xi.y;
									dp.y = eta_t * Xi.y / dt;
									if (ep.x < 0.0f) { ep.y = dp.y = 0.0f; }
									if (abs(ep.y) > mu * ep.x) { ep.y = mu * sign(ep.x, ep.y);		dp.y = 0.0f; }
									d_ep.y[couple] = ep.y;
									Fcol.y = ep.y + dp.y;
									//////////////////////���[�J��y///////////////////////////////

									//////////////////////���[�J��z///////////////////////////////
									ep.z += kt * Xi.z;
									dp.z = eta_t * Xi.z / dt;
									if (ep.x < 0.0f) { ep.z = dp.z = 0.0f; }
									if (abs(ep.z) > mu * ep.x) { ep.z = mu * sign(ep.x, ep.z);		dp.z = 0.0f; }
									d_ep.z[couple] = ep.z;
									Fcol.z = ep.z + dp.z;
									//////////////////////���[�J��z///////////////////////////////


									//////////////////////���[���h///////////////////////////////
									d_Ftotal.x[i] -= n.x * Fcol.x - n.y * Fcol.y / L_2 - n.x * n.z * Fcol.z / L_2;
									d_Ftotal.y[i] -= n.y * Fcol.x + n.x * Fcol.y / L_2 - n.y * n.z * Fcol.z / L_2;
									d_Ftotal.z[i] -= n.z * Fcol.x + Fcol.z * L_2;

									d_Torque.x[i] -= n.x * Tr - (-n.y * Fcol.z / L_2 + n.x * n.z * Fcol.y / L_2) * Di * 0.5f;
									d_Torque.y[i] -= n.y * Tr - (n.x * Fcol.z / L_2 + n.y * n.z * Fcol.y / L_2) * Di * 0.5f;
									d_Torque.z[i] -= n.z * Tr + Fcol.y * L_2 * Di * 0.5f;
									//////////////////////���[���h///////////////////////////////

								}
							}
							else {//�ڐG���Ă��Ȃ���,ep��0,pair��-1�ɂ��Ă����B
								for (int k = 0; k < NCP; k++) {
									int kiNPWLLNCP = k + i * NCP;
									if (d_pair[kiNPWLLNCP] == j) {
										d_ep.x[kiNPWLLNCP] = 0.0f;
										d_ep.y[kiNPWLLNCP] = 0.0f;
										d_ep.z[kiNPWLLNCP] = 0.0f;
										d_pair[kiNPWLLNCP] = -2;
										break;
									}
								}
							}
						}
					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

}


void DEMPS::ColForce() {
	//printf_s("ColForce start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = (nP);	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_ColForce << <blocks_nP, threads >> > (nP, nPSLD, nPWLL, d_D, d_Typ, d_Pos, d_Vel, d_Ftotal, d_Omega, d_Torque, d_ep, d_pair, m, eta_n, eta_t, kn, kt, mu, dt, MINc, DBinv, nBx, nBxy, d_bfst, d_blst, d_nxt);
	//CHECK(hipDeviceSynchronize());

	//printf_s("ColForce finished!\n\n");
}


__global__ void d_update(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque, const real usmax,
	const real m, const real dt, const treal3 MINc, const treal3 MAXc, const treal3 G, const real PCL_DST, const real I)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= nP) { return; }

	if (d_Typ[i] != SLD) { return; }//���̑��x�X�V

		/*treal3 Vtmp;
		Vtmp.x = d_Vel.x[i];		Vtmp.y = d_Vel.y[i];		Vtmp.z = d_Vel.z[i];*/

	d_Vel.x[i] += d_Ftotal.x[i] * dt / m;
	d_Vel.y[i] += d_Ftotal.y[i] * dt / m + G.y * dt;
	d_Vel.z[i] += d_Ftotal.z[i] * dt / m;

	treal3 Utmp;
	Utmp.x = d_Vel.x[i];	Utmp.y = d_Vel.y[i];	Utmp.z = d_Vel.z[i];
	real U = Utmp.x * Utmp.x + Utmp.y * Utmp.y + Utmp.z * Utmp.z;
	U = sqrt(U);
	if (U > usmax) {
		Utmp.x *= usmax / U;	Utmp.y *= usmax / U;	Utmp.z *= usmax / U;
		d_Vel.x[i] = Utmp.x;		d_Vel.y[i] = Utmp.y;		d_Vel.z[i] = Utmp.z;
	}

	/*d_Pos.x[i] += 0.5f * (Vtmp.x + d_Vel.x[i]) * dt;//�O�X�e�b�v�ƌ��݂̑��x�̕��ϒl�������ړ�������
	d_Pos.y[i] += 0.5f * (Vtmp.y + d_Vel.y[i]) * dt;
	d_Pos.z[i] += 0.5f * (Vtmp.z + d_Vel.z[i]) * dt;*/

	d_Pos.x[i] += d_Vel.x[i] * dt;
	d_Pos.y[i] += d_Vel.y[i] * dt;
	d_Pos.z[i] += d_Vel.z[i] * dt;

	if (d_Pos.x[i] > MAXc.x - 3.0f * PCL_DST) { d_Typ[i] = GST; }
	else if (d_Pos.y[i] > MAXc.y - 3.0f * PCL_DST) { d_Typ[i] = GST; }
	else if (d_Pos.z[i] > MAXc.z - 3.0f * PCL_DST) { d_Typ[i] = GST; }
	else if (d_Pos.x[i] < MINc.x + 3.0f * PCL_DST) { d_Typ[i] = GST; }
	else if (d_Pos.y[i] < MINc.y + 3.0f * PCL_DST) { d_Typ[i] = GST; }
	else if (d_Pos.z[i] < MINc.z + 3.0f * PCL_DST) { d_Typ[i] = GST; }

	d_Omega.x[i] += d_Torque.x[i] * dt / I;
	d_Omega.y[i] += d_Torque.y[i] * dt / I;
	d_Omega.z[i] += d_Torque.z[i] * dt / I;

	d_Ftotal.x[i] = 0.0f;
	d_Ftotal.y[i] = 0.0f;
	d_Ftotal.z[i] = 0.0f;

	d_Torque.x[i] = 0.0f;
	d_Torque.y[i] = 0.0f;
	d_Torque.z[i] = 0.0f;

}


void DEMPS::update() {
	//printf_s("update start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_update << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Ftotal, d_Omega, d_Torque, usmax, m, dt, MINc, MAXc, G, PCL_DST, I);
	//CHECK(hipDeviceSynchronize());

	//printf_s("update finished!\n\n");
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////DEM_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif


#if Multi_flg
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////Multi_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#if 0
//���W�Ԋ�
__global__ void d_SLD_FLD(const int nP, real* d_D, real* d_Dns, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque, const real dt,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= nP) { return; }
	char Typ = d_Typ[i];
	if (Typ != SLD) { return; }

	treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
	treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
	treal3 Omegai;	Omegai.x = d_Omega.x[i];	Omegai.y = d_Omega.y[i];	Omegai.z = d_Omega.z[i];
	real Di = d_D[i];
	real dnsi = d_Dns[Typ];


	int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
	int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
	int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;

	for (int jz = iz - 1; jz <= iz + 1; jz++) {
		for (int jy = iy - 1; jy <= iy + 1; jy++) {
			for (int jx = ix - 1; jx <= ix + 1; jx++) {
				int jb = jz * nBxy + jy * nBx + jx;
				int j = d_bfst[jb];
				if (j == -1) continue;
				for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
					if (j != i) {
						char typ = d_Typ[j];
						if (typ == FLD) {
							treal3 vs = { 0.0f };//���[�J���ő̑��x
							treal3 vl = { 0.0f };//���[�J�����̑��x

							treal3 r_delta_i;//���[�J����]���x Tgl * Omega

							treal3 dist;
							real L;
							real L_2;
							treal3 n;

							treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
							treal3 Velj;		Velj.x = d_Vel.x[j];		Velj.y = d_Vel.y[j];		Velj.z = d_Vel.z[j];
							real Dj = d_D[j];
							real dnsj = d_Dns[typ];

							dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;

							L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);

							n.x = dist.x / L;//l   
							n.y = dist.y / L;//m
							n.z = dist.z / L;//n

							L_2 = sqrt(n.x * n.x + n.y * n.y);//�����]���̒��g

							treal3 Fpint = { 0.0f };//�t��=>�ő̗̂�
							real area = 0.5f * (Di + Dj);
							if (L < area) {//�ڐG����
								real w = WEI(L, area) / wtotal;
								if (L_2 == 0) {//�W���o�����b�N�΍�
									r_delta_i.x = -Omegai.z;
									r_delta_i.y = Omegai.y;
									r_delta_i.z = Omegai.x;

									vs.x = -Veli.z;
									vs.y = Veli.y + r_delta_i.z * L;
									vs.z = Veli.x - r_delta_i.y * L;
									vl.x = -Velj.z;
									vl.y = Velj.y;
									vl.z = Velj.x;

									Fpint.x = w * (dnsi * vs.x - dnsj * vl.x) * dt;
									Fpint.y = w * (dnsi * vs.y - dnsj * vl.y) * dt;
									Fpint.z = w * (dnsi * vs.z - dnsj * vl.z) * dt;

									d_Ftotal.x[i] -= Fpint.z;
									d_Ftotal.y[i] -= Fpint.y;
									d_Ftotal.z[i] -= -Fpint.x;

									d_Torque.x[i] -= Fpint.y * L;
									d_Torque.y[i] -= -Fpint.z * L;

									//atomicAdd(&d_Ftotal.x[j], Fpint.z);
									//atomicAdd(&d_Ftotal.y[j], Fpint.y);
									//atomicAdd(&d_Ftotal.z[j], -Fpint.x);
									d_Ftotal.x[j] += Fpint.z;
									d_Ftotal.y[j] += Fpint.y;
									d_Ftotal.z[j] += -Fpint.x;

								}
								else {//�ʏ�
									r_delta_i.x = (n.x * Omegai.x + n.y * Omegai.y + n.z * Omegai.z);
									r_delta_i.y = (-n.y * Omegai.x / L_2 + n.x * Omegai.y / L_2);
									r_delta_i.z = (-n.x * n.z * Omegai.x / L_2 - n.y * n.z * Omegai.y / L_2 + L_2 * Omegai.z);

									vs.x = (n.x * Veli.x + n.y * Veli.y + n.z * Veli.z);
									vs.y = (-n.y * Veli.x / L_2 + n.x * Veli.y / L_2) + r_delta_i.z * L;
									vs.z = (-n.x * n.z * Veli.x / L_2 - n.y * n.z * Veli.y / L_2 + Veli.z * L_2) - r_delta_i.y * L;
									vl.x = (n.x * Velj.x + n.y * Velj.y + n.z * Velj.z);
									vl.y = (-n.y * Velj.x / L_2 + n.x * Velj.y / L_2);
									vl.z = (-n.x * n.z * Velj.x / L_2 - n.y * n.z * Velj.y / L_2 + Velj.z * L_2);

									Fpint.x = w * (dnsi * vs.x - dnsj * vl.x) * dt;
									Fpint.y = w * (dnsi * vs.y - dnsj * vl.y) * dt;
									Fpint.z = w * (dnsi * vs.z - dnsj * vl.z) * dt;

									d_Ftotal.x[i] -= n.x * Fpint.x - n.y * Fpint.y / L_2 - n.x * n.z * Fpint.z / L_2;
									d_Ftotal.y[i] -= n.y * Fpint.x + n.x * Fpint.y / L_2 - n.y * n.z * Fpint.z / L_2;
									d_Ftotal.z[i] -= n.z * Fpint.x + Fpint.z * L_2;

									d_Torque.x[i] -= - (-n.y * Fpint.z / L_2 + n.x * n.z * Fpint.y / L_2) * L;
									d_Torque.y[i] -= - (n.x * Fpint.z / L_2 + n.y * n.z * Fpint.y / L_2) * L;
									d_Torque.z[i] -= Fpint.y * L_2 * L;

									//atomicAdd(&d_Ftotal.x[j], n.x * Fpint.x - n.y * Fpint.y / L_2 - n.x * n.z * Fpint.z / L_2);
									//atomicAdd(&d_Ftotal.y[j], n.y * Fpint.x + n.x * Fpint.y / L_2 - n.y * n.z * Fpint.z / L_2);
									//atomicAdd(&d_Ftotal.z[j], n.z * Fpint.x + Fpint.z * L_2);
									d_Ftotal.x[j] += n.x * Fpint.x - n.y * Fpint.y / L_2 - n.x * n.z * Fpint.z / L_2;
									d_Ftotal.y[j] += n.y * Fpint.x + n.x * Fpint.y / L_2 - n.y * n.z * Fpint.z / L_2;
									d_Ftotal.z[j] += -n.z * Fpint.x + Fpint.z * L_2;

								}
							}

						}

					}
					j = d_nxt[j];
					if (j == -1) break;
				}//���qi�̋ߖT���qj�̃��[�v�I��
			}
		}
	}

}


//��]���݂̉^���ʌ����@���삿�イ
__global__ void d_SLD_FLD(const int nP, real* d_D, real* d_Dns, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque, const real dt, const real Vol_SLD, const real Vol_FLD,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	char Typ = d_Typ[i];

	if (Typ == SLD) {
		treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
		treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
		treal3 vec2;		vec2.x = d_Vel.x[i];		vec2.y = d_Vel.y[i];		vec2.z = d_Vel.z[i];
		real Di = d_D[i];
		//real dnsi = d_Dns[Typ];
		real mi = d_Dns[Typ] * Vol_SLD;

		int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
		int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
		int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;

		//���̃��[�v
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							char typ = d_Typ[j];
							if (typ == FLD) {
								treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
								treal3 dist;	 dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;
								real L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
								real Dj = d_D[j];
								real area = 0.5f * (Di + Dj);
								if (L < area) {
									treal3 Velj;		Velj.x = d_Vel.x[j];		Velj.y = d_Vel.y[j];		Velj.z = d_Vel.z[j];
									treal3 vec;
									vec.x = Veli.x;
									vec.y = Veli.y;
									vec.z = Veli.z;
									//real dnsj = d_Dns[typ];
									real mj = d_Dns[typ] * Vol_FLD;
									real fDT = (vec.x - Velj.x) * dist.x + (vec.y - Velj.y) * dist.y + (vec.z - Velj.z) * dist.z;
									if (fDT > 0.0f) {//���������̏ꍇ�̏����H
										fDT *= mj / (mi + mj) / L;
										vec2.x -= dist.x * fDT;	vec2.y -= dist.y * fDT;	vec2.z -= dist.z * fDT;
									}
								}

							}

						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
		d_Acc.x[i] = vec2.x;	d_Acc.y[i] = vec2.y;	d_Acc.z[i] = vec2.z;
	}

	else if (Typ == FLD) {
		treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
		treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
		treal3 vec2;		vec2.x = d_Vel.x[i];		vec2.y = d_Vel.y[i];		vec2.z = d_Vel.z[i];
		real Di = d_D[i];
		//real dnsi = d_Dns[Typ];
		real mi = d_Dns[Typ] * Vol_FLD;

		int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
		int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
		int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;

		//���̃��[�v
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							char typ = d_Typ[j];
							if (typ == SLD) {
								treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
								treal3 dist;	 dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;
								real L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
								real Dj = d_D[j];
								real area = 0.5f * (Di + Dj);
								if (L < area) {
									treal3 Velj;
									Velj.x = d_Vel.x[j];
									Velj.y = d_Vel.y[j]; 
									Velj.z = d_Vel.z[j]; 
									//real dnsj = d_Dns[typ];
									real mj = d_Dns[typ] * Vol_SLD;
									real fDT = (Veli.x - Velj.x) * dist.x + (Veli.y - Velj.y) * dist.y + (Veli.z - Velj.z) * dist.z;
									if (fDT > 0.0f) {//���������̏ꍇ�̏����H
										fDT *= mj / (mi + mj) / L;
										vec2.x -= dist.x * fDT;	vec2.y -= dist.y * fDT;	vec2.z -= dist.z * fDT;
									}
								}

							}

						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
		d_Acc.x[i] = vec2.x;	d_Acc.y[i] = vec2.y;	d_Acc.z[i] = vec2.z;
	}

}


void DEMPS::SLD_FLD() {
	//printf_s("SLD_FLD start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = (nP);	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_SLD_FLD << <blocks_nP, threads >> > (nP, d_D, d_Dns, d_Typ, d_Pos, d_Vel, d_Acc, d_Ftotal, d_Omega, d_Torque, dt, Vol_SLD, Vol_FLD, MINc, DBinv, nBx, nBxy, d_bfst, d_blst, d_nxt);
	CHECK(hipDeviceSynchronize());

	//printf_s("SLD_FLD finished!\n\n");
}

#else
//chkcol�Ɠ��������Ōv�Z
__global__ void d_SLD_FLD(const int nP, real* d_D, real* d_Dns, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque, const real dt, const real Vol_SLD, const real Vol_FLD,
	const treal3 MINc, const real DBinv, const int nBx, const int nBxy, const int* d_bfst, const int* d_blst, const int* d_nxt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	char Typ = d_Typ[i];

	if (Typ == SLD) {
		treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
		treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
		treal3 vec2;		vec2.x = d_Vel.x[i];		vec2.y = d_Vel.y[i];		vec2.z = d_Vel.z[i];
		//treal3 Omegai;	Omegai.x = d_Omega.x[i];	Omegai.y = d_Omega.y[i];	Omegai.z = d_Omega.z[i];
		real Di = d_D[i];
		//real dnsi = d_Dns[Typ];
		real mi = d_Dns[Typ] * Vol_SLD;

		int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
		int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
		int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;

		//���̃��[�v
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							char typ = d_Typ[j];
							if (typ == FLD) {
								treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
								treal3 dist;	 dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;
								real L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
								real Dj = d_D[j];
								real area = 0.5f * (Di + Dj);
								if (L < area) {
									treal3 Velj;		Velj.x = d_Vel.x[j];		Velj.y = d_Vel.y[j];		Velj.z = d_Vel.z[j];
									treal3 vec;		
									vec.x = Veli.x;// +(dist.y * Omegai.z - dist.z * Omegai.y);
									vec.y = Veli.y;// +(dist.z * Omegai.x - dist.x * Omegai.z);
									vec.z = Veli.z;// +(dist.x * Omegai.y - dist.y * Omegai.x);
									//real dnsj = d_Dns[typ];
									real mj = d_Dns[typ] * Vol_FLD;
									real fDT = (vec.x - Velj.x) * dist.x + (vec.y - Velj.y) * dist.y + (vec.z - Velj.z) * dist.z;
									if (fDT > 0.0f) {//���������̏ꍇ�̏����H
										fDT *= mj / (mi + mj) / L;
										vec2.x -= dist.x * fDT;	vec2.y -= dist.y * fDT;	vec2.z -= dist.z * fDT;
									}
								}

							}

						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
		d_Acc.x[i] = vec2.x;	d_Acc.y[i] = vec2.y;	d_Acc.z[i] = vec2.z;
	}

	else if (Typ == FLD) {
		treal3 Posi;	Posi.x = d_Pos.x[i];	Posi.y = d_Pos.y[i];	Posi.z = d_Pos.z[i];
		treal3 Veli;		Veli.x = d_Vel.x[i];		Veli.y = d_Vel.y[i];		Veli.z = d_Vel.z[i];
		treal3 vec2;		vec2.x = d_Vel.x[i];		vec2.y = d_Vel.y[i];		vec2.z = d_Vel.z[i];
		real Di = d_D[i];
		//real dnsi = d_Dns[Typ];
		real mi= d_Dns[Typ] * Vol_FLD;

		int ix = (int)((Posi.x - MINc.x) * DBinv) + 1;
		int iy = (int)((Posi.y - MINc.y) * DBinv) + 1;
		int iz = (int)((Posi.z - MINc.z) * DBinv) + 1;

		//���̃��[�v
		for (int jz = iz - 1; jz <= iz + 1; jz++) {
			for (int jy = iy - 1; jy <= iy + 1; jy++) {
				for (int jx = ix - 1; jx <= ix + 1; jx++) {
					int jb = jz * nBxy + jy * nBx + jx;
					int j = d_bfst[jb];
					if (j == -1) continue;
					for (;;) {//���qi�̋ߖT���qj�̃��[�v�J�n
						if (j != i) {
							char typ = d_Typ[j];
							if (typ == SLD) {
								treal3 Posj;	Posj.x = d_Pos.x[j];	Posj.y = d_Pos.y[j];	Posj.z = d_Pos.z[j];
								treal3 dist;	 dist.x = Posj.x - Posi.x;   dist.y = Posj.y - Posi.y;   dist.z = Posj.z - Posi.z;
								real L = sqrt(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
								real Dj = d_D[j];
								real area = 0.5f * (Di + Dj);
								if (L < area) {
									//treal3 Omegaj;	Omegaj.x = d_Omega.x[j];	Omegaj.y = d_Omega.y[j];	Omegaj.z = d_Omega.z[j];
									treal3 Velj;
									Velj.x = d_Vel.x[j]; //- (dist.y * Omegaj.z - dist.z * Omegaj.y);//dist�̕������t�ɂȂ��Ă�H����
									Velj.y = d_Vel.y[j]; //-(dist.z * Omegaj.x - dist.x * Omegaj.z);
									Velj.z = d_Vel.z[j]; //-(dist.x * Omegaj.y - dist.y * Omegaj.x);
									//real dnsj = d_Dns[typ];
									real mj = d_Dns[typ] * Vol_SLD;
									real fDT = (Veli.x - Velj.x) * dist.x + (Veli.y - Velj.y) * dist.y + (Veli.z - Velj.z) * dist.z;
									if (fDT > 0.0f) {//���������̏ꍇ�̏����H
									fDT *= mj / (mi + mj) / L;
									vec2.x -= dist.x * fDT;	vec2.y -= dist.y * fDT;	vec2.z -= dist.z * fDT;
									}
								}

							}

						}
						j = d_nxt[j];
						if (j == -1) break;
					}//���qi�̋ߖT���qj�̃��[�v�I��
				}
			}
		}
		d_Acc.x[i] = vec2.x;	d_Acc.y[i] = vec2.y;	d_Acc.z[i] = vec2.z;
	}

}
#endif

void DEMPS::SLD_FLD() {
	//printf_s("SLD_FLD start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = (nP);	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_SLD_FLD << <blocks_nP, threads >> > (nP, d_D, d_Dns, d_Typ, d_Pos, d_Vel, d_Acc, d_Ftotal, d_Omega, d_Torque, dt, Vol_SLD, Vol_FLD, MINc, DBinv, nBx, nBxy, d_bfst, d_blst, d_nxt);
	CHECK(hipDeviceSynchronize());

	//printf_s("SLD_FLD finished!\n\n");
}

#if 0
__global__ void d_SF_update(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Ftotal, areal3 d_Omega, areal3 d_Torque, const real m, const real dt, const real I)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }

	char Typ = d_Typ[i];
	if (Typ == SLD) {
		treal3 acc;
		acc.x = d_Ftotal.x[i] / Dns_SLD;
		acc.y = d_Ftotal.y[i] / Dns_SLD;
		acc.z = d_Ftotal.z[i] / Dns_SLD;//���� or ���x ?

		d_Vel.x[i] += acc.x * dt;
		d_Vel.y[i] += acc.y * dt;
		d_Vel.z[i] += acc.z * dt;

		d_Pos.x[i] += acc.x * dt * dt;
		d_Pos.y[i] += acc.y * dt * dt;
		d_Pos.z[i] += acc.z * dt * dt;

		d_Omega.x[i] += d_Torque.x[i] * dt / I;
		d_Omega.y[i] += d_Torque.y[i] * dt / I;
		d_Omega.z[i] += d_Torque.z[i] * dt / I;

	}
	else if (Typ == FLD) {
		treal3 acc;
		acc.x = d_Ftotal.x[i] / Dns_FLD;
		acc.y = d_Ftotal.y[i] / Dns_FLD;
		acc.z = d_Ftotal.z[i] / Dns_FLD;

		d_Vel.x[i] += acc.x * dt;
		d_Vel.y[i] += acc.y * dt;
		d_Vel.z[i] += acc.z * dt;

		d_Pos.x[i] += acc.x * dt * dt;
		d_Pos.y[i] += acc.y * dt * dt;
		d_Pos.z[i] += acc.z * dt * dt;
	}

	d_Ftotal.x[i] = 0.0f;
	d_Ftotal.y[i] = 0.0f;
	d_Ftotal.z[i] = 0.0f;

	d_Torque.x[i] = 0.0f;
	d_Torque.y[i] = 0.0f;
	d_Torque.z[i] = 0.0f;

}


void DEMPS::SF_update() {
	//printf_s("update start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_SF_update << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Ftotal, d_Omega, d_Torque, m, dt, I);
	//CHECK(hipDeviceSynchronize());

	//printf_s("update finished!\n\n");
}
#else
__global__ void d_SF_update(const int nP, char* d_Typ, areal3 d_Pos, areal3 d_Vel, areal3 d_Acc)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nP) { return; }
	char Typ = d_Typ[i];
	if ((Typ == SLD) || (Typ == FLD)) {
		d_Vel.x[i] = d_Acc.x[i];		d_Vel.y[i] = d_Acc.y[i];		d_Vel.z[i] = d_Acc.z[i];
		d_Acc.x[i] = d_Acc.y[i] = d_Acc.z[i] = 0.0f;
	}
	
}


void DEMPS::SF_update() {
	//printf_s("update start!\n");
	////////////////cuda�X���b�h�ݒ�/////////////////////
	dim3 threads(THREADS, 1, 1);
	int TOTAL_THREADS = nBxyz;	int BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nBxyz(BLOCKS, 1, 1);
	TOTAL_THREADS = (nP);	BLOCKS = TOTAL_THREADS / THREADS + 1;
	dim3 blocks_nP(BLOCKS, 1, 1);
	//////////////////////////////////////////////
	d_SF_update << <blocks_nP, threads >> > (nP, d_Typ, d_Pos, d_Vel, d_Acc);
	//CHECK(hipDeviceSynchronize());

	//printf_s("update finished!\n\n");
}
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////Multi_Function/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif


void DEMPS::DevicetoHost() {
	//printf_s("DevicetoHost start!\n");
	CHECK(hipMemcpy(Typ, d_Typ, sizeof(char) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Pos.x, d_Pos.x, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Pos.y, d_Pos.y, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Pos.z, d_Pos.z, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Vel.x, d_Vel.x, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Vel.y, d_Vel.y, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Vel.z, d_Vel.z, sizeof(real) * nP, hipMemcpyDeviceToHost));

#if 0 //update�ŏ��������Ă��܂��Ă�@drill�̂�Q�l�ɗv�ύX
	CHECK(hipMemcpy(Ftotal.x, d_Ftotal.x, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Ftotal.y, d_Ftotal.y, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Ftotal.z, d_Ftotal.z, sizeof(real) * nP, hipMemcpyDeviceToHost));
#endif


#if MPS_flg
	CHECK(hipMemcpy(Prs, d_Prs, sizeof(real) * nP, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(pav, d_pav, sizeof(real) * nP, hipMemcpyDeviceToHost));

	CHECK(hipMemcpy(TypM, d_TypM, sizeof(char) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(PosM.x, d_PosM.x, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(PosM.y, d_PosM.y, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(PosM.z, d_PosM.z, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(VelM.x, d_VelM.x, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(VelM.y, d_VelM.y, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(VelM.z, d_VelM.z, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(PrsM, d_PrsM, sizeof(real) * (nP * NumMRR), hipMemcpyDeviceToHost));
#endif


	//printf_s("DevicetoHost finished!\n\n");
}





void DEMPS::ClcDEMPS() {
	iF = 0;
	TIM = 0.0f;
	outtime = 0.0f;
	OPT_FQC = 1.0f;

#if MPS_flg
	MkBkt();
	Surface_Edge();//�Ǔ������Ƃ��͖��X�e�b�v���s�ł���悤�Ɉړ������邱�ƁI
#endif

	WrtDatWLL();
	WrtDat();
	printf_s("\n");

	iF++;
	OPT_FQC = 0.0f;


	while (1) {

		//////////////////////�o��///////////////////////////
		if (outtime >= output_time) {
			outtime -= output_time;

			DevicetoHost();

			WrtDat();
			printf_s("Time = %f\n\n", TIM);
			iF++;
			if (TIM >= FIN_TIM) {
				WrtDat2();
				break;
			}
			OPT_FQC = 0.0f;
		}
		//////////////////////�o��///////////////////////////

#if MPS_flg
		//////////////////////MPS///////////////////////////
		MkBkt();

		ResetMRR();
		GenMRR_nonslip();
		MkBkt_MRR();

		VscTrm();

		UpPcl1();

		MkBkt();

		ChkCol();

		MkBkt();//�����H

		MkPrs();

		ResetMRR();
		GenMRR_nonslip();
		MkBkt_MRR();

		PrsGrdTrm();

		UpPcl2();
		//////////////////////MPS///////////////////////////
#endif


#if Multi_flg
		//////////////////////���ݍ�p///////////////////////////

		SLD_FLD();

		SF_update();

		//////////////////////���ݍ�p///////////////////////////
#endif


#if DEM_flg
		//////////////////////DEM///////////////////////////
		MkBkt();

		ColForce();

		update();
		//////////////////////DEM///////////////////////////
#endif


		outtime += dt;
		TIM += dt;
		OPT_FQC += 1.0f;
		//printf_s("time=%f\n", TIM);
	}


}


void DEMPS::memory_free() {
	free(Pos.x); free(Pos.y); free(Pos.z);
	free(Vel.x); free(Vel.y); free(Vel.z);
	free(Omega.x); free(Omega.y); free(Omega.z);
	free(Ftotal.x); free(Ftotal.y); free(Ftotal.z);
	free(Torque.x); free(Torque.y); free(Torque.z);
	free(Typ);
	free(ep.x); free(ep.y); free(ep.z);
	free(D);
	free(pair);

	hipFree(d_Pos.x); hipFree(d_Pos.y); hipFree(d_Pos.z);
	hipFree(d_Vel.x); hipFree(d_Vel.y); hipFree(d_Vel.z);
	hipFree(d_Omega.x); hipFree(d_Omega.y); hipFree(d_Omega.z);
	hipFree(d_Ftotal.x); hipFree(d_Ftotal.y); hipFree(d_Ftotal.z);
	hipFree(d_Torque.x); hipFree(d_Torque.y); hipFree(d_Torque.z);
	hipFree(d_Typ);
	hipFree(d_ep.x); hipFree(d_ep.y); hipFree(d_ep.z);
	hipFree(d_D);
	hipFree(d_pair);


	free(Acc.x);	free(Acc.y);	free(Acc.z);
	free(Prs);
	free(pav);
	free(TypM);
	free(PosM.x);		free(PosM.y);		free(PosM.z);
	free(VelM.x);		free(VelM.y);		free(VelM.z);
	free(PrsM);
	free(Dns);
	free(WLLSE);
	free(WLLVec.x);		free(WLLVec.y);		free(WLLVec.z);

	hipFree(d_Acc.x);	hipFree(d_Acc.y);	hipFree(d_Acc.z);
	hipFree(d_Prs);
	hipFree(d_pav);
	hipFree(d_TypM);
	hipFree(d_PosM.x);		hipFree(d_PosM.y);		hipFree(d_PosM.z);
	hipFree(d_VelM.x);		hipFree(d_VelM.y);		hipFree(d_VelM.z);
	hipFree(d_PrsM);
	hipFree(d_Dns);
	hipFree(d_WLLSE);
	hipFree(d_WLLVec.x);		hipFree(d_WLLVec.y);		hipFree(d_WLLVec.z);
}