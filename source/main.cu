#include "class.cuh"

int main(int argc, char** argv) {
	std::chrono::system_clock::time_point  start, end; // �^�� auto �ŉ�
	start = std::chrono::system_clock::now(); // �v���J�n����

	if (Multi_flg == 1) { printf_s("��������͂����s���܂�.\n\n"); }
	else{ 
		if (DEM_flg == 1) { printf_s("DEM�����s���܂�.\n\n"); }
		if (MPS_flg == 1) { printf_s("MPS�����s���܂�.\n\n"); }
	}
	
	printf_s("�g�p�\�ȍő�X���b�h���F%d\n", omp_get_max_threads());
#pragma omp parallel
	for (int i = 0; i < omp_get_max_threads(); i++) {
		printf_s("Hello!! CPU Thread %d\n", i);
	}
	printf_s("\n");

	DEMPS obj;

	obj.RdDat();
	printf_s("RdDat finished!\n\n");

	obj.AlcBkt();
	printf_s("AlkBkt finished!\n\n");

	obj.SetPara();
	printf_s("SetPara finished!\n\n");

	obj.ClcDEMPS();

	printf_s("Coaculation finished!!!\n\n");

	end = std::chrono::system_clock::now();  // �v���I������
	double elapsed = std::chrono::duration_cast<std::chrono::seconds>(end - start).count(); //�����ɗv�������Ԃ�b�ɕϊ�
	std::cout << elapsed << " second" << std::endl;

	int t = elapsed;
	int h = t / 3600;   t %= 3600;
	int m = t / 60;     t %= 60;
	int s = t;
	std::cout << h << "h " << m << "m " << s << "s " << std::endl;

	obj.memory_free();

	return 0;
}
